/*
 * Copyright (c) 2023, NVIDIA CORPORATION & AFFILIATES.
 *
 * SPDX-License-Identifier: BSD-3-Clause
 */

#include <hip/hip_runtime_api.h> // hipMalloc, hipMemcpy, etc.
#include <hip/hip_complex.h>        // hipDoubleComplex
#include <custatevec.h>       // custatevecInitializeStateVector
#include <stdio.h>            // printf
#include <stdlib.h>           // EXIT_FAILURE

#include "helper.hpp"         // HANDLE_ERROR, HANDLE_CUDA_ERROR

int main(void) {

    const int nIndexBits = 3;
    const int svSize = (1 << nIndexBits);

    hipDoubleComplex h_sv[svSize];

    hipDoubleComplex h_sv_result[] = {{ 1.0, 0.0}, { 0.0, 0.0}, { 0.0, 0.0}, { 0.0, 0.0},
                                     { 0.0, 0.0}, { 0.0, 0.0}, { 0.0, 0.0}, { 0.0, 0.0}};

    hipDoubleComplex *d_sv;
    HANDLE_CUDA_ERROR( hipMalloc((void**)&d_sv, svSize * sizeof(hipDoubleComplex)) );

    // populate the device memory with junk values (for illustrative purpose only)
    HANDLE_CUDA_ERROR( hipMemset(d_sv, 0x7F, svSize * sizeof(hipDoubleComplex)) );

    //----------------------------------------------------------------------------------------------

    // custatevec handle initialization
    custatevecHandle_t handle;
    HANDLE_ERROR( custatevecCreate(&handle) );

    // initialize the state vector
    HANDLE_ERROR( custatevecInitializeStateVector(
                  handle, d_sv, HIP_C_64F, nIndexBits, CUSTATEVEC_STATE_VECTOR_TYPE_ZERO) );

    // destroy handle
    HANDLE_ERROR( custatevecDestroy(handle) );

    //----------------------------------------------------------------------------------------------

    HANDLE_CUDA_ERROR( hipMemcpy(h_sv, d_sv, svSize * sizeof(hipDoubleComplex),
                       hipMemcpyDeviceToHost) );

    bool correct = true;
    for (int i = 0; i < svSize; i++) {
        if (!almost_equal(h_sv[i], h_sv_result[i])) {
            correct = false;
            break;
        }
    }

    HANDLE_CUDA_ERROR( hipFree(d_sv) );

    if (correct) {
        printf("initialize_sv example PASSED\n");
        return EXIT_SUCCESS;
    }
    else {
        printf("initialize_sv example FAILED: wrong result\n");
        return EXIT_FAILURE;
    }

}
