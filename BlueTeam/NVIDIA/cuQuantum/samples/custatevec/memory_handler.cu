#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2021-2023, NVIDIA CORPORATION & AFFILIATES.
 *
 * SPDX-License-Identifier: BSD-3-Clause
 */

#include <hip/hip_runtime_api.h> // hipMalloc, hipMemcpy, etc.
#include <hip/hip_complex.h>        // hipDoubleComplex
#include <custatevec.h>       // custatevecApplyMatrix
#include <string.h>           // strcpy
#include <stdio.h>            // printf
#include <stdlib.h>           // EXIT_FAILURE

#include "helper.hpp"         // HANDLE_ERROR, HANDLE_CUDA_ERROR

#define SUPPORTS_MEMORY_POOL ( __CUDACC_VER_MAJOR__ > 11 || (__CUDACC_VER_MAJOR__ == 11 && __CUDACC_VER_MINOR__ >= 2))
#if SUPPORTS_MEMORY_POOL

// upon success, this function should return 0, otherwise a nonzero value
int myMemPoolAlloc(void* ctx, void** ptr, size_t size, hipStream_t stream) {
    hipMemPool_t& pool = *static_cast<hipMemPool_t*>(ctx);
    hipError_t status = hipMallocFromPoolAsync(ptr, size, pool, stream);
    return (int)status;
}

// upon success, this function should return 0, otherwise a nonzero value
int myMemPoolFree(void*, void* ptr, size_t, hipStream_t stream) {
    hipError_t status = hipFreeAsync(ptr, stream);
    return (int)status;
}

int main(void) {
    // state vector
    const int nIndexBits   = 3;
    const int nSvSize      = (1 << nIndexBits);

    hipDoubleComplex h_sv[] = {{ 0.48, 0.0}, { 0.36, 0.0}, { 0.64, 0.0}, { 0.48, 0.0}, 
                              { 0.0,  0.0}, { 0.0,  0.0}, { 0.0,  0.0}, { 0.0,  0.0}};

    //----------------------------------------------------------------------------------------------
    // gates
    const int adjoint = 0;
    const custatevecMatrixLayout_t layout = CUSTATEVEC_MATRIX_LAYOUT_ROW;

    // Hadamard gate
    const int hTargets[] = {2};
    const uint32_t hNTargets = 1;
    const double Rsqrt2 = 1. / std::sqrt(2.);
    hipDoubleComplex hGate[] = {{Rsqrt2, 0.0}, {Rsqrt2, 0.0},
                               {Rsqrt2, 0.0}, {-Rsqrt2, 0.0}};

    // control-SWAP gate
    const int swapTargets[] = {0, 1};
    const uint32_t swapNTargets = 2;
    const int swapControls[] = {2};
    const uint32_t swapNControls = 1;
    hipDoubleComplex swapGate[] = {{1.0, 0.0}, {0.0, 0.0}, {0.0, 0.0}, {0.0, 0.0},
                                  {0.0, 0.0}, {0.0, 0.0}, {1.0, 0.0}, {0.0, 0.0},
                                  {0.0, 0.0}, {1.0, 0.0}, {0.0, 0.0}, {0.0, 0.0},
                                  {0.0, 0.0}, {0.0, 0.0}, {0.0, 0.0}, {1.0, 0.0}};

    // observable
    const int basisBits[] = {2};
    const uint32_t nBasisBits = 1;
    hipDoubleComplex observable[] = {{1.0, 0.0}, {0.0, 0.0},
                                    {0.0, 0.0}, {0.0, 0.0}};

    //----------------------------------------------------------------------------------------------
    // device configuration
    int deviceId;
    HANDLE_CUDA_ERROR( hipGetDevice(&deviceId) );

    hipError_t status;
    int isMemPoolSupported;
    status = hipDeviceGetAttribute(&isMemPoolSupported, hipDeviceAttributeMemoryPoolsSupported, deviceId);
    if (status != hipSuccess || !isMemPoolSupported) {
        printf("memory handler example WAIVED: CUDA Memory pools is not supported.\n");
        return EXIT_SUCCESS;
    }

    hipMemPool_t memPool;
    HANDLE_CUDA_ERROR( hipDeviceGetDefaultMemPool(&memPool, deviceId) );

    // avoid shrinking the pool 
    uint64_t threshold = UINT64_MAX;
    hipMemPoolSetAttribute(memPool, hipMemPoolAttrReleaseThreshold, &threshold);

    hipStream_t stream;
    HANDLE_CUDA_ERROR( hipStreamCreate(&stream) );

    //----------------------------------------------------------------------------------------------
    // data transfer of state vector
    hipDoubleComplex *d_sv;
    HANDLE_CUDA_ERROR( hipMallocAsync((void**)&d_sv, nSvSize * sizeof(hipDoubleComplex), stream) );

    HANDLE_CUDA_ERROR( hipMemcpyAsync(d_sv, h_sv, nSvSize * sizeof(hipDoubleComplex), 
                                       hipMemcpyHostToDevice, stream) );

    //----------------------------------------------------------------------------------------------
    // custatevec handle initialization
    custatevecHandle_t handle;
    HANDLE_ERROR( custatevecCreate(&handle) );
    HANDLE_ERROR( custatevecSetStream(handle, stream) );

    // device memory handler
    custatevecDeviceMemHandler_t handler;
    handler.ctx = &memPool;
    handler.device_alloc = myMemPoolAlloc;
    handler.device_free = myMemPoolFree;
    strcpy(handler.name, "mempool");
    HANDLE_ERROR( custatevecSetDeviceMemHandler(handle, &handler) );

    // apply Hadamard gate
    HANDLE_ERROR( custatevecApplyMatrix(
                  handle, d_sv, HIP_C_64F, nIndexBits, hGate, HIP_C_64F,
                  layout, adjoint, hTargets, hNTargets, nullptr, nullptr, 0, 
                  CUSTATEVEC_COMPUTE_DEFAULT, nullptr, 0) );

    // apply control-SWAP gate
    HANDLE_ERROR( custatevecApplyMatrix(
                  handle, d_sv, HIP_C_64F, nIndexBits, swapGate, HIP_C_64F,
                  layout, adjoint, swapTargets, swapNTargets, swapControls, nullptr, swapNControls, 
                  CUSTATEVEC_COMPUTE_DEFAULT, nullptr, 0) );

    // apply Hadamard gate
    HANDLE_ERROR( custatevecApplyMatrix(
                  handle, d_sv, HIP_C_64F, nIndexBits, hGate, HIP_C_64F,
                  layout, adjoint, hTargets, hNTargets, nullptr, nullptr, 0, 
                  CUSTATEVEC_COMPUTE_DEFAULT, nullptr, 0) );

    // compute expectation
    double expectationValue;
    HANDLE_ERROR( custatevecComputeExpectation(
                  handle, d_sv, HIP_C_64F, nIndexBits, &expectationValue, HIP_R_64F, nullptr,
                  observable, HIP_C_64F, layout, basisBits, nBasisBits,
                  CUSTATEVEC_COMPUTE_DEFAULT, nullptr, 0) );

    HANDLE_CUDA_ERROR( hipStreamSynchronize(stream) );

    // destroy handle
    HANDLE_ERROR( custatevecDestroy(handle) );

    //----------------------------------------------------------------------------------------------

    // release device memory and stream
    HANDLE_CUDA_ERROR( hipFreeAsync(d_sv, stream) );
    HANDLE_CUDA_ERROR( hipStreamDestroy(stream) );

    double expectationValueResult = 0.9608;
    bool correct = almost_equal(expectationValue, expectationValueResult);
    if (correct) {
        printf("memory_handler example PASSED\n");
        return EXIT_SUCCESS;
    }
    else {
        printf("memory_handler example FAILED: wrong result\n");
        return EXIT_FAILURE;
    }
}

#else
int main(void) {
    printf("memory_handler example WAIVED : This example uses CUDA's built-in stream-ordered memory allocator, which requires CUDA 11.2+.\n");
    return EXIT_SUCCESS;
}
#endif