/*
 * Copyright (c) 2023, NVIDIA CORPORATION & AFFILIATES.
 *
 * SPDX-License-Identifier: BSD-3-Clause
 */

#include <hip/hip_runtime_api.h> // hipMalloc, hipMemcpy, etc.
#include <hip/hip_complex.h>        // hipDoubleComplex
#include <custatevec.h>       // custatevecApplyMatrixBatched
#include <stdio.h>            // printf
#include <stdlib.h>           // EXIT_FAILURE

#include "helper.hpp"         // HANDLE_ERROR, HANDLE_CUDA_ERROR

int main(void) {

    const int nSVs       = 2;
    const int nIndexBits = 3;
    const int nSvSize    = (1 << nIndexBits);
    const int svStride   = nSvSize;
    const int nTargets   = 1;
    const int nControls  = 2;
    const int adjoint    = 0;

    const int targets[]  = {2};
    const int controls[] = {0, 1};

    const int nMatrices = 2;
    const int matrixIndices[] = {1, 0};

    // 2 state vectors are allocated contiguously in single memory chunk.
    hipDoubleComplex h_svs[]        = {{ 0.0, 0.0}, { 0.0, 0.1}, { 0.1, 0.1}, { 0.1, 0.2}, 
                                      { 0.2, 0.2}, { 0.3, 0.3}, { 0.3, 0.4}, { 0.4, 0.5},
                                      { 0.0, 0.0}, { 0.0, 0.1}, { 0.1, 0.1}, { 0.1, 0.2}, 
                                      { 0.2, 0.2}, { 0.3, 0.3}, { 0.3, 0.4}, { 0.4, 0.5}};
    hipDoubleComplex h_svs_result[] = {{ 0.0, 0.0}, { 0.0, 0.1}, { 0.1, 0.1}, { 0.1, 0.2}, 
                                      { 0.2, 0.2}, { 0.3, 0.3}, { 0.3, 0.4}, {-0.4,-0.5},
                                      { 0.0, 0.0}, { 0.0, 0.1}, { 0.1, 0.1}, { 0.4, 0.5}, 
                                      { 0.2, 0.2}, { 0.3, 0.3}, { 0.3, 0.4}, { 0.1, 0.2}};
    // 2 gate matrices are allocated contiguously in single memory chunk.
    hipDoubleComplex matrices[] = {{0.0, 0.0}, {1.0, 0.0},
                                  {1.0, 0.0}, {0.0, 0.0},
                                  {1.0, 0.0}, {0.0, 0.0},
                                  {0.0, 0.0}, {-1.0, 0.0}};

    hipDoubleComplex *d_svs;
    HANDLE_CUDA_ERROR( hipMalloc((void**)&d_svs, nSVs * svStride * sizeof(hipDoubleComplex)) );
 
    HANDLE_CUDA_ERROR( hipMemcpy(d_svs, h_svs, nSVs * svStride * sizeof(hipDoubleComplex),
                       hipMemcpyHostToDevice) );

    //---------------------------------------------------------------------------------------------

    // custatevec handle initialization
    custatevecHandle_t handle;
    HANDLE_ERROR( custatevecCreate(&handle) );

    void* extraWorkspace = nullptr;
    size_t extraWorkspaceSizeInBytes = 0;

    // check the size of external workspace
    HANDLE_ERROR( custatevecApplyMatrixBatchedGetWorkspaceSize(
                  handle, HIP_C_64F, nIndexBits, nSVs, svStride,
                  CUSTATEVEC_MATRIX_MAP_TYPE_MATRIX_INDEXED, matrixIndices, matrices, HIP_C_64F,
                  CUSTATEVEC_MATRIX_LAYOUT_ROW, adjoint, nMatrices, nTargets, nControls,
                  CUSTATEVEC_COMPUTE_64F, &extraWorkspaceSizeInBytes) );

    // allocate external workspace if necessary
    if (extraWorkspaceSizeInBytes > 0)
        HANDLE_CUDA_ERROR( hipMalloc(&extraWorkspace, extraWorkspaceSizeInBytes) );

    // apply gate
    HANDLE_ERROR( custatevecApplyMatrixBatched(
                  handle, d_svs, HIP_C_64F, nIndexBits, nSVs, svStride,
                  CUSTATEVEC_MATRIX_MAP_TYPE_MATRIX_INDEXED, matrixIndices, matrices, HIP_C_64F,
                  CUSTATEVEC_MATRIX_LAYOUT_ROW, adjoint, nMatrices, targets, nTargets, controls,
                  nullptr, nControls, CUSTATEVEC_COMPUTE_64F, extraWorkspace,
                  extraWorkspaceSizeInBytes) );

    // destroy handle
    HANDLE_ERROR( custatevecDestroy(handle) );

    //---------------------------------------------------------------------------------------------

    HANDLE_CUDA_ERROR( hipMemcpy(h_svs, d_svs, nSVs * svStride * sizeof(hipDoubleComplex),
                       hipMemcpyDeviceToHost) );

    bool correct = true;
    for (int i = 0; i < nSVs * svStride; i++) {
        if (!almost_equal(h_svs[i], h_svs_result[i])) {
            correct = false;
            break;
        }
    }

    HANDLE_CUDA_ERROR( hipFree(d_svs) );
    if (extraWorkspaceSizeInBytes)
        HANDLE_CUDA_ERROR( hipFree(extraWorkspace) );

    if (correct) {
        printf("batched_gate_application example PASSED\n");
        return EXIT_SUCCESS;
    }
    else {
        printf("batched_gate_application example FAILED: wrong result\n");
        return EXIT_FAILURE;
    }

}
