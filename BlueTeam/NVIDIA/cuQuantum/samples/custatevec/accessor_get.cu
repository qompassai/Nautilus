/*
 * Copyright (c) 2021-2023, NVIDIA CORPORATION & AFFILIATES.
 *
 * SPDX-License-Identifier: BSD-3-Clause
 */

#include <hip/hip_runtime_api.h> // hipMalloc, hipMemcpy, etc.
#include <hip/hip_complex.h>        // hipDoubleComplex
#include <custatevec.h>       // custatevecApplyMatrix
#include <stdio.h>            // printf
#include <stdlib.h>           // EXIT_FAILURE

#include "helper.hpp"         // HANDLE_ERROR, HANDLE_CUDA_ERROR

int main(void) {

    const int nIndexBits = 3;
    const int nSvSize    = (1 << nIndexBits);

    const int bitOrderingLen = 2;
    const int bitOrdering[]  = {2, 1};

    const int maskLen         = 1;
    const int maskBitString[] = {1};
    const int maskOrdering[]  = {0};

    const int bufferSize  = 3;
    const int accessBegin = 1;
    const int accessEnd   = 4;

    hipDoubleComplex h_sv[]        = {{ 0.0, 0.0}, { 0.0, 0.1}, { 0.1, 0.1}, { 0.1, 0.2},
                                     { 0.2, 0.2}, { 0.3, 0.3}, { 0.3, 0.4}, { 0.4, 0.5}};
    hipDoubleComplex buffer[]        = {{ 0.0, 0.0}, { 0.0, 0.0}, { 0.0, 0.0}};
    hipDoubleComplex buffer_result[] = {{ 0.3, 0.3}, { 0.1, 0.2}, { 0.4, 0.5}};

    custatevecAccessorDescriptor_t accessor;

    hipDoubleComplex *d_sv;
    HANDLE_CUDA_ERROR( hipMalloc((void**)&d_sv, nSvSize * sizeof(hipDoubleComplex)) );

    HANDLE_CUDA_ERROR( hipMemcpy(d_sv, h_sv, nSvSize * sizeof(hipDoubleComplex),
                       hipMemcpyHostToDevice) );

    //----------------------------------------------------------------------------------------------

    // custatevec handle initialization
    custatevecHandle_t handle;
    HANDLE_ERROR( custatevecCreate(&handle) );

    void* extraWorkspace = nullptr;
    size_t extraWorkspaceSizeInBytes = 0;

    // create accessor and check the size of external workspace
    HANDLE_ERROR( custatevecAccessorCreateView(
                  handle, d_sv, HIP_C_64F, nIndexBits, &accessor, bitOrdering, bitOrderingLen,
                  maskBitString, maskOrdering, maskLen, &extraWorkspaceSizeInBytes) );

    // allocate external workspace if necessary
    if (extraWorkspaceSizeInBytes > 0)
        HANDLE_CUDA_ERROR( hipMalloc(&extraWorkspace, extraWorkspaceSizeInBytes) );

    // set external workspace
    HANDLE_ERROR( custatevecAccessorSetExtraWorkspace(
                  handle, accessor, extraWorkspace, extraWorkspaceSizeInBytes) );

    // get state vector components
    HANDLE_ERROR( custatevecAccessorGet(
                  handle, accessor, buffer, accessBegin, accessEnd) );

    // destroy descriptor and handle
    HANDLE_ERROR( custatevecAccessorDestroy(accessor) );
    HANDLE_ERROR( custatevecDestroy(handle) );

    //----------------------------------------------------------------------------------------------

    HANDLE_CUDA_ERROR( hipMemcpy(h_sv, d_sv, nSvSize * sizeof(hipDoubleComplex),
                       hipMemcpyDeviceToHost) );

    bool correct = true;
    for (int i = 0; i < bufferSize; i++) {
        if (!almost_equal(buffer[i], buffer_result[i])) {
            correct = false;
            break;
        }
    }

    HANDLE_CUDA_ERROR( hipFree(d_sv) );
    if (extraWorkspaceSizeInBytes)
        HANDLE_CUDA_ERROR( hipFree(extraWorkspace) );

    if (correct) {
        printf("accessor_get example PASSED\n");
        return EXIT_SUCCESS;
    }
    else {
        printf("accessor_get example FAILED: wrong result\n");
        return EXIT_FAILURE;
    }

}
