/*
 * Copyright (c) 2021-2023, NVIDIA CORPORATION & AFFILIATES.
 *
 * SPDX-License-Identifier: BSD-3-Clause
 */

#include <hip/hip_runtime_api.h> // hipMalloc, hipMemcpy, etc.
#include <hip/hip_complex.h>        // hipDoubleComplex
#include <custatevec.h>       // custatevecApplyMatrix
#include <stdio.h>            // printf
#include <stdlib.h>           // EXIT_FAILURE

#include "helper.hpp"         // HANDLE_ERROR, HANDLE_CUDA_ERROR

int main(void) {

    const int nIndexBits = 3;
    const int nSvSize    = (1 << nIndexBits);
    const int nBasisBits = 1;
    const int maskLen    = 0;
    const int adjoint    = 0;

    const int basisBits[] = {2};

    hipDoubleComplex h_sv[]        = {{ 0.0, 0.0}, { 0.0, 0.1}, { 0.1, 0.1}, { 0.1, 0.2},
                                     { 0.2, 0.2}, { 0.3, 0.3}, { 0.3, 0.4}, { 0.4, 0.5}};
    hipDoubleComplex h_sv_result[] = {{ 0.0, 0.0}, { 0.0, 0.1}, { 0.1, 0.1}, { 0.1, 0.2},
                                     { 0.2,-0.2}, { 0.3,-0.3}, { 0.4,-0.3}, { 0.5,-0.4}};
    hipDoubleComplex diagonals[] = {{1.0, 0.0}, {0.0, -1.0}};

    hipDoubleComplex *d_sv;
    HANDLE_CUDA_ERROR( hipMalloc((void**)&d_sv, nSvSize * sizeof(hipDoubleComplex)) );

    HANDLE_CUDA_ERROR( hipMemcpy(d_sv, h_sv, nSvSize * sizeof(hipDoubleComplex),
                       hipMemcpyHostToDevice) );

    //----------------------------------------------------------------------------------------------

    // custatevec handle initialization
    custatevecHandle_t handle;
    HANDLE_ERROR( custatevecCreate(&handle) );

    void* extraWorkspace = nullptr;
    size_t extraWorkspaceSizeInBytes = 0;

    // check the size of external workspace
    HANDLE_ERROR( custatevecApplyGeneralizedPermutationMatrixGetWorkspaceSize(
                  handle, HIP_C_64F, nIndexBits, nullptr, diagonals, HIP_C_64F, basisBits,
                  nBasisBits, maskLen, &extraWorkspaceSizeInBytes) );

    // allocate external workspace if necessary
    if (extraWorkspaceSizeInBytes > 0)
        HANDLE_CUDA_ERROR( hipMalloc(&extraWorkspace, extraWorkspaceSizeInBytes) );

    // apply matrix
    HANDLE_ERROR( custatevecApplyGeneralizedPermutationMatrix(
                  handle, d_sv, HIP_C_64F, nIndexBits, nullptr, diagonals, HIP_C_64F, adjoint,
                  basisBits, nBasisBits, nullptr, nullptr, maskLen, extraWorkspace,
                  extraWorkspaceSizeInBytes) );

    // destroy handle
    HANDLE_ERROR( custatevecDestroy(handle) );

    //----------------------------------------------------------------------------------------------

    HANDLE_CUDA_ERROR( hipMemcpy(h_sv, d_sv, nSvSize * sizeof(hipDoubleComplex),
                       hipMemcpyDeviceToHost) );

    bool correct = true;
    for (int i = 0; i < nSvSize; i++) {
        if (!almost_equal(h_sv[i], h_sv_result[i])) {
            correct = false;
            break;
        }
    }

    HANDLE_CUDA_ERROR( hipFree(d_sv) );
    if (extraWorkspaceSizeInBytes)
        HANDLE_CUDA_ERROR( hipFree(extraWorkspace) );

    if (correct) {
        printf("diagonal_matrix example PASSED\n");
        return EXIT_SUCCESS;
    }
    else {
        printf("diagonal_matrix example FAILED: wrong result\n");
        return EXIT_FAILURE;
    }

}
