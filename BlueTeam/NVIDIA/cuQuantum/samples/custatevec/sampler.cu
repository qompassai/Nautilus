/*
 * Copyright (c) 2021-2023, NVIDIA CORPORATION & AFFILIATES.
 *
 * SPDX-License-Identifier: BSD-3-Clause
 */

#include <hip/hip_runtime_api.h> // hipMalloc, hipMemcpy, etc.
#include <hip/hip_complex.h>        // hipDoubleComplex
#include <custatevec.h>       // custatevecApplyMatrix
#include <stdio.h>            // printf
#include <stdlib.h>           // EXIT_FAILURE

#include "helper.hpp"         // HANDLE_ERROR, HANDLE_CUDA_ERROR

int main(void) {

    const int nIndexBits = 3;
    const int nSvSize    = (1 << nIndexBits);
    const int nMaxShots  = 5;
    const int nShots     = 5;

    const int bitStringLen  = 2;
    const int bitOrdering[] = {0, 1};

    custatevecIndex_t bitStrings[nShots];
    custatevecIndex_t bitStrings_result[] = {0b00, 0b01, 0b10, 0b11, 0b11};

    hipDoubleComplex h_sv[]        = {{ 0.0, 0.0}, { 0.0, 0.1}, { 0.1, 0.1}, { 0.1, 0.2}, 
                                     { 0.2, 0.2}, { 0.3, 0.3}, { 0.3, 0.4}, { 0.4, 0.5}};

    // In real appliction, random numbers in range [0, 1) will be used.
    const double randnums[] = {0.1, 0.8, 0.4, 0.6, 0.2};

    custatevecSamplerDescriptor_t sampler;

    hipDoubleComplex *d_sv;
    HANDLE_CUDA_ERROR( hipMalloc((void**)&d_sv, nSvSize * sizeof(hipDoubleComplex)) );

    HANDLE_CUDA_ERROR( hipMemcpy(d_sv, h_sv, nSvSize * sizeof(hipDoubleComplex), 
                       hipMemcpyHostToDevice) );

    //----------------------------------------------------------------------------------------------

    // custatevec handle initialization
    custatevecHandle_t handle;
    HANDLE_ERROR( custatevecCreate(&handle) );

    void* extraWorkspace = nullptr;
    size_t extraWorkspaceSizeInBytes = 0;

    // create sampler and check the size of external workspace
    HANDLE_ERROR( custatevecSamplerCreate(
                  handle, d_sv, HIP_C_64F, nIndexBits, &sampler, nMaxShots, 
                  &extraWorkspaceSizeInBytes) );
    
    // allocate external workspace if necessary
    if (extraWorkspaceSizeInBytes > 0)
        HANDLE_CUDA_ERROR( hipMalloc(&extraWorkspace, extraWorkspaceSizeInBytes) );
    
    // sample preprocess
    HANDLE_ERROR( custatevecSamplerPreprocess(
                  handle, sampler, extraWorkspace, extraWorkspaceSizeInBytes) );
    
    // sample bit strings
    HANDLE_ERROR( custatevecSamplerSample(
                  handle, sampler, bitStrings, bitOrdering, bitStringLen, randnums, nShots, 
                  CUSTATEVEC_SAMPLER_OUTPUT_ASCENDING_ORDER) );

    // destroy descriptor and handle
    HANDLE_ERROR( custatevecSamplerDestroy(sampler) );
    HANDLE_ERROR( custatevecDestroy(handle) );

    //----------------------------------------------------------------------------------------------

    HANDLE_CUDA_ERROR( hipMemcpy(h_sv, d_sv, nSvSize * sizeof(hipDoubleComplex),
                       hipMemcpyDeviceToHost) );

    bool correct = true;
    for (int i = 0; i < nShots; i++) {
        if (bitStrings[i] != bitStrings_result[i]) {
            correct = false;
            break;
        }
    }

    HANDLE_CUDA_ERROR( hipFree(d_sv) );
    if (extraWorkspaceSizeInBytes)
        HANDLE_CUDA_ERROR( hipFree(extraWorkspace) );

    if (correct) {
        printf("sampler example PASSED\n");
        return EXIT_SUCCESS;
    }
    else {
        printf("sampler example FAILED: wrong result\n");
        return EXIT_FAILURE;
    }
}
