#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2021-2023, NVIDIA CORPORATION & AFFILIATES.
 *
 * SPDX-License-Identifier: BSD-3-Clause
 */

#include <hip/hip_runtime_api.h> // hipMalloc, hipMemcpy, etc.
#include <hip/hip_complex.h>        // hipDoubleComplex
#include <custatevec.h>       // custatevecApplyMatrix
#include <stdio.h>            // printf
#include <stdlib.h>           // EXIT_FAILURE
#include <cmath>              // acos

#include "helper.hpp"         // HANDLE_ERROR, HANDLE_CUDA_ERROR

int main(void) {

    const int nIndexBits = 3;
    const int nSvSize    = (1 << nIndexBits);
    const int nTargets   = 1;
    const int nControls  = 1;

    const int targets[]  = {2};
    const int controls[] = {1};
    const int controlBitValues[] = {1};

    const double pi = std::acos(-1.0);

    const custatevecPauli_t paulis[] = {CUSTATEVEC_PAULI_Z};

    hipDoubleComplex h_sv[]        = {{ 0.0, 0.0}, { 0.0, 0.1}, { 0.1, 0.1}, { 0.1, 0.2}, 
                                     { 0.2, 0.2}, { 0.3, 0.3}, { 0.3, 0.4}, { 0.4, 0.5}};
    hipDoubleComplex h_sv_result[] = {{ 0.0, 0.0}, { 0.0, 0.1}, {-0.1, 0.1}, {-0.2, 0.1}, 
                                     { 0.2, 0.2}, { 0.3, 0.3}, { 0.4,-0.3}, { 0.5,-0.4}};

    hipDoubleComplex *d_sv;
    HANDLE_CUDA_ERROR( hipMalloc((void**)&d_sv, nSvSize * sizeof(hipDoubleComplex)) );

    HANDLE_CUDA_ERROR( hipMemcpy(d_sv, h_sv, nSvSize * sizeof(hipDoubleComplex), 
                       hipMemcpyHostToDevice) );

    //----------------------------------------------------------------------------------------------

    // custatevec handle initialization
    custatevecHandle_t handle;
    HANDLE_ERROR( custatevecCreate(&handle) );

    // apply Pauli operator
    HANDLE_ERROR( custatevecApplyPauliRotation(
                  handle, d_sv, HIP_C_64F, nIndexBits, pi / 2.0, paulis, targets, nTargets, 
                  controls, controlBitValues, nControls) );

    // destroy handle
    HANDLE_ERROR( custatevecDestroy(handle) );

    //----------------------------------------------------------------------------------------------

    HANDLE_CUDA_ERROR( hipMemcpy(h_sv, d_sv, nSvSize * sizeof(hipDoubleComplex),
                       hipMemcpyDeviceToHost) );

    bool correct = true;
    for (int i = 0; i < nSvSize; i++) {
        if ( !almost_equal(h_sv[i], h_sv_result[i]) ) {
            correct = false;
             break;
        }
    }

    HANDLE_CUDA_ERROR( hipFree(d_sv) );

    if (correct) {
        printf("exponential_pauli example PASSED\n");
        return EXIT_SUCCESS;
    }
    else {
        printf("exponential_pauli example FAILED: wrong result\n");
        return EXIT_FAILURE;
    }

}
