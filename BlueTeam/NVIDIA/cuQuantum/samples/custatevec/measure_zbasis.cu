/*
 * Copyright (c) 2021-2023, NVIDIA CORPORATION & AFFILIATES.
 *
 * SPDX-License-Identifier: BSD-3-Clause
 */

#include <hip/hip_runtime_api.h> // hipMalloc, hipMemcpy, etc.
#include <hip/hip_complex.h>        // hipDoubleComplex
#include <custatevec.h>       // custatevecApplyMatrix
#include <stdio.h>            // printf
#include <stdlib.h>           // EXIT_FAILURE

#include "helper.hpp"         // HANDLE_ERROR, HANDLE_CUDA_ERROR

int main(void) {

    const int nIndexBits   = 3;
    const int nSvSize      = (1 << nIndexBits);
    const int nBasisBits   = 3;

    const int basisBits[] = {0, 1, 2};

    int parity;
    const int parity_result = 0;

    // In real appliction, random number in range [0, 1) will be used.
    const double randnum = 0.2;

    hipDoubleComplex h_sv[]        = {{ 0.0, 0.0}, { 0.0, 0.1}, { 0.3, 0.4}, { 0.1, 0.2}, 
                                     { 0.2, 0.2}, { 0.3, 0.3}, { 0.1, 0.1}, { 0.4, 0.5}};
    hipDoubleComplex h_sv_result[] = {{ 0.0, 0.0}, { 0.0, 0.0}, { 0.0, 0.0}, { 0.2, 0.4}, 
                                     { 0.0, 0.0}, { 0.6, 0.6}, { 0.2, 0.2}, { 0.0, 0.0}};

    hipDoubleComplex *d_sv;
    HANDLE_CUDA_ERROR( hipMalloc((void**)&d_sv, nSvSize * sizeof(hipDoubleComplex)) );

    HANDLE_CUDA_ERROR( hipMemcpy(d_sv, h_sv, nSvSize * sizeof(hipDoubleComplex), 
                       hipMemcpyHostToDevice) );

    //----------------------------------------------------------------------------------------------

    // custatevec handle initialization
    custatevecHandle_t handle;
    HANDLE_ERROR( custatevecCreate(&handle) );

    // measurement on z basis
    HANDLE_ERROR( custatevecMeasureOnZBasis(
                  handle, d_sv, HIP_C_64F, nIndexBits, &parity, basisBits, nBasisBits, 
                  randnum, CUSTATEVEC_COLLAPSE_NORMALIZE_AND_ZERO) );

    // destroy handle
    HANDLE_ERROR( custatevecDestroy(handle) );

    //----------------------------------------------------------------------------------------------

    HANDLE_CUDA_ERROR( hipMemcpy(h_sv, d_sv, nSvSize * sizeof(hipDoubleComplex),
                       hipMemcpyDeviceToHost) );

    bool correct = true;
    for (int i = 0; i < nSvSize; i++) {
        if (!almost_equal(h_sv[i], h_sv_result[i])) {
            correct = false;
            break;
        }
    }

    if (parity != parity_result) {
       correct = false;
    }

    HANDLE_CUDA_ERROR( hipFree(d_sv) );

    if (correct) {
        printf("measure_zbasis example PASSED\n");
        return EXIT_SUCCESS;
    }
    else {
        printf("measure_zbasis example FAILED: wrong result\n");
        return EXIT_FAILURE;
    }

}

