#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2021-2023, NVIDIA CORPORATION & AFFILIATES.
 *
 * SPDX-License-Identifier: BSD-3-Clause
 */

#include <hip/hip_runtime_api.h> // hipMalloc, hipMemcpy, etc.
#include <hip/hip_complex.h>        // hipDoubleComplex
#include <custatevec.h>       // custatevecTestMatrixType
#include <stdio.h>            // printf
#include <stdlib.h>           // EXIT_FAILURE

#include "helper.hpp"         // HANDLE_ERROR, HANDLE_CUDA_ERROR

double runTestMatrixType(custatevecHandle_t       handle,
                         custatevecMatrixType_t   matrixType,
                         const void*              matrix,
                         hipDataType           matrixDataType,
                         custatevecMatrixLayout_t layout,
                         const uint32_t           nTargets,
                         const int32_t            adjoint,
                         custatevecComputeType_t  computeType) {

    double residualNorm;

    void* extraWorkspace = nullptr;
    size_t extraWorkspaceSizeInBytes = 0;

    // check the size of external workspace
    HANDLE_ERROR( custatevecTestMatrixTypeGetWorkspaceSize(
                  handle, matrixType, matrix, matrixDataType, layout,
                  nTargets, adjoint, computeType, &extraWorkspaceSizeInBytes) );

    // allocate external workspace if necessary
    if (extraWorkspaceSizeInBytes > 0)
        HANDLE_CUDA_ERROR( hipMalloc(&extraWorkspace, extraWorkspaceSizeInBytes) );

    // execute testing
    HANDLE_ERROR( custatevecTestMatrixType(
                  handle, &residualNorm, matrixType, matrix, matrixDataType, layout,
                  nTargets, adjoint, computeType, extraWorkspace, extraWorkspaceSizeInBytes) );

    HANDLE_CUDA_ERROR( hipDeviceSynchronize() );

    if (extraWorkspaceSizeInBytes)
        HANDLE_CUDA_ERROR( hipFree(extraWorkspace) );

    return residualNorm;
}

int main(void) {

    const int nTargets = 1;
    const int adjoint = 0;

    // unitary and Hermitian matrix
    const double Rsqrt2 = 1. / std::sqrt(2.);
    hipDoubleComplex matrix[] = {{0.5, 0.0}, {Rsqrt2, -0.5},
                                {Rsqrt2, 0.5}, {-0.5, 0.0}};

    //----------------------------------------------------------------------------------------------

    // custatevec handle initialization
    custatevecHandle_t handle;
    HANDLE_ERROR( custatevecCreate(&handle) );

    hipDataType matrixDataType = HIP_C_64F;
    custatevecMatrixLayout_t layout = CUSTATEVEC_MATRIX_LAYOUT_ROW;
    custatevecComputeType_t computeType = CUSTATEVEC_COMPUTE_DEFAULT;

    double unitaryResidualNorm = runTestMatrixType(handle, CUSTATEVEC_MATRIX_TYPE_UNITARY, matrix,
                                                   matrixDataType, layout, nTargets, adjoint,
                                                   computeType) ;

    double hermiteResidualNorm = runTestMatrixType(handle, CUSTATEVEC_MATRIX_TYPE_HERMITIAN, matrix,
                                                   matrixDataType, layout, nTargets, adjoint,
                                                   computeType) ;

    // destroy handle
    HANDLE_ERROR( custatevecDestroy(handle) );

    //----------------------------------------------------------------------------------------------

    bool correct = true;

    correct &= almost_equal(unitaryResidualNorm, 0.);
    correct &= almost_equal(hermiteResidualNorm, 0.);

    if (correct) {
        printf("test_matrix_type example PASSED\n");
        return EXIT_SUCCESS;
    }
    else {
        printf("test_matrix_type example FAILED: wrong result\n");
        return EXIT_FAILURE;
    }

}