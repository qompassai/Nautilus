#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2021-2024, NVIDIA CORPORATION & AFFILIATES.
 *
 * SPDX-License-Identifier: BSD-3-Clause
 */

// Sphinx: #1

#include <stdlib.h>
#include <stdio.h>

#include <unordered_map>
#include <vector>
#include <cassert>

#include <hip/hip_runtime.h>
#include <cutensornet.h>


#define HANDLE_ERROR(x)                                           \
{ const auto err = x;                                             \
  if( err != CUTENSORNET_STATUS_SUCCESS )                         \
  { printf("Error: %s in line %d\n", cutensornetGetErrorString(err), __LINE__); \
    fflush(stdout);                                               \
  }                                                               \
};

#define HANDLE_CUDA_ERROR(x)                                      \
{ const auto err = x;                                             \
  if( err != hipSuccess )                                        \
  { printf("CUDA Error: %s in line %d\n", hipGetErrorString(err), __LINE__); \
    fflush(stdout);                                               \
  }                                                               \
};


struct GPUTimer
{
    GPUTimer(hipStream_t stream): stream_(stream)
    {
        HANDLE_CUDA_ERROR(hipEventCreate(&start_));
        HANDLE_CUDA_ERROR(hipEventCreate(&stop_));
    }

    ~GPUTimer()
    {
        HANDLE_CUDA_ERROR(hipEventDestroy(start_));
        HANDLE_CUDA_ERROR(hipEventDestroy(stop_));
    }

    void start()
    {
        HANDLE_CUDA_ERROR(hipEventRecord(start_, stream_));
    }

    float seconds()
    {
        HANDLE_CUDA_ERROR(hipEventRecord(stop_, stream_));
        HANDLE_CUDA_ERROR(hipEventSynchronize(stop_));
        float time;
        HANDLE_CUDA_ERROR(hipEventElapsedTime(&time, start_, stop_));
        return time * 1e-3;
    }

    private:
    hipEvent_t start_, stop_;
    hipStream_t stream_;
};


int main()
{
   static_assert(sizeof(size_t) == sizeof(int64_t), "Please build this sample on a 64-bit architecture!");

   bool verbose = true;

   // Check cuTensorNet version
   const size_t cuTensornetVersion = cutensornetGetVersion();
   if(verbose)
      printf("cuTensorNet version: %ld\n", cuTensornetVersion);

   // Set GPU device
   int numDevices {0};
   HANDLE_CUDA_ERROR( hipGetDeviceCount(&numDevices) );
   const int deviceId = 0;
   HANDLE_CUDA_ERROR( hipSetDevice(deviceId) );
   hipDeviceProp_t prop;
   HANDLE_CUDA_ERROR( hipGetDeviceProperties(&prop, deviceId) );

   if(verbose) {
      printf("===== device info ======\n");
      printf("GPU-local-id:%d\n", deviceId);
      printf("GPU-name:%s\n", prop.name);
      printf("GPU-clock:%d\n", prop.clockRate);
      printf("GPU-memoryClock:%d\n", prop.memoryClockRate);
      printf("GPU-nSM:%d\n", prop.multiProcessorCount);
      printf("GPU-major:%d\n", prop.major);
      printf("GPU-minor:%d\n", prop.minor);
      printf("========================\n");
   }

   typedef float floatType;
   hipDataType typeData = HIP_R_32F;
   cutensornetComputeType_t typeCompute = CUTENSORNET_COMPUTE_32F;

   if(verbose)
      printf("Included headers and defined data types\n");

   // Sphinx: #2
   /**********************
   * Computing: R_{k,l} = A_{a,b,c,d,e,f} B_{b,g,h,e,i,j} C_{m,a,g,f,i,k} D_{l,c,h,d,j,m}
   **********************/

   constexpr int32_t numInputs = 4;

   // Create vectors of tensor modes
   std::vector<int32_t> modesA{'a','b','c','d','e','f'};
   std::vector<int32_t> modesB{'b','g','h','e','i','j'};
   std::vector<int32_t> modesC{'m','a','g','f','i','k'};
   std::vector<int32_t> modesD{'l','c','h','d','j','m'};
   std::vector<int32_t> modesR{'k','l'};

   // Set mode extents
   std::unordered_map<int32_t, int64_t> extent;
   extent['a'] = 16;
   extent['b'] = 16;
   extent['c'] = 16;
   extent['d'] = 16;
   extent['e'] = 16;
   extent['f'] = 16;
   extent['g'] = 16;
   extent['h'] = 16;
   extent['i'] = 16;
   extent['j'] = 16;
   extent['k'] = 16;
   extent['l'] = 16;
   extent['m'] = 16;

   // Create a vector of extents for each tensor
   std::vector<int64_t> extentA;
   for (auto mode : modesA)
      extentA.push_back(extent[mode]);
   std::vector<int64_t> extentB;
   for (auto mode : modesB)
      extentB.push_back(extent[mode]);
   std::vector<int64_t> extentC;
   for (auto mode : modesC)
      extentC.push_back(extent[mode]);
   std::vector<int64_t> extentD;
   for (auto mode : modesD)
      extentD.push_back(extent[mode]);
   std::vector<int64_t> extentR;
   for (auto mode : modesR)
      extentR.push_back(extent[mode]);

   if(verbose)
      printf("Defined tensor network, modes, and extents\n");

   // Sphinx: #3
   /**********************
   * Allocating data
   **********************/

   size_t elementsA = 1;
   for (auto mode : modesA)
      elementsA *= extent[mode];
   size_t elementsB = 1;
   for (auto mode : modesB)
      elementsB *= extent[mode];
   size_t elementsC = 1;
   for (auto mode : modesC)
      elementsC *= extent[mode];
   size_t elementsD = 1;
   for (auto mode : modesD)
      elementsD *= extent[mode];
   size_t elementsR = 1;
   for (auto mode : modesR)
      elementsR *= extent[mode];

   size_t sizeA = sizeof(floatType) * elementsA;
   size_t sizeB = sizeof(floatType) * elementsB;
   size_t sizeC = sizeof(floatType) * elementsC;
   size_t sizeD = sizeof(floatType) * elementsD;
   size_t sizeR = sizeof(floatType) * elementsR;
   if(verbose)
      printf("Total GPU memory used for tensor storage: %.2f GiB\n",
             (sizeA + sizeB + sizeC + sizeD + sizeR) / 1024. /1024. / 1024);

   void* rawDataIn_d[numInputs];
   void* R_d;
   HANDLE_CUDA_ERROR( hipMalloc((void**) &rawDataIn_d[0], sizeA) );
   HANDLE_CUDA_ERROR( hipMalloc((void**) &rawDataIn_d[1], sizeB) );
   HANDLE_CUDA_ERROR( hipMalloc((void**) &rawDataIn_d[2], sizeC) );
   HANDLE_CUDA_ERROR( hipMalloc((void**) &rawDataIn_d[3], sizeD) );
   HANDLE_CUDA_ERROR( hipMalloc((void**) &R_d, sizeR));

   floatType *A = (floatType*) malloc(sizeof(floatType) * elementsA);
   floatType *B = (floatType*) malloc(sizeof(floatType) * elementsB);
   floatType *C = (floatType*) malloc(sizeof(floatType) * elementsC);
   floatType *D = (floatType*) malloc(sizeof(floatType) * elementsD);
   floatType *R = (floatType*) malloc(sizeof(floatType) * elementsR);

   if (A == NULL || B == NULL || C == NULL || D == NULL || R == NULL)
   {
      printf("Error: Host memory allocation failed!\n");
      return -1;
   }

   /*******************
   * Initialize data
   *******************/

   memset(R, 0, sizeof(floatType) * elementsR);
   for (uint64_t i = 0; i < elementsA; i++)
      A[i] = ((floatType) rand()) / RAND_MAX;
   for (uint64_t i = 0; i < elementsB; i++)
      B[i] = ((floatType) rand()) / RAND_MAX;
   for (uint64_t i = 0; i < elementsC; i++)
      C[i] = ((floatType) rand()) / RAND_MAX;
   for (uint64_t i = 0; i < elementsD; i++)
      D[i] = ((floatType) rand()) / RAND_MAX;

   HANDLE_CUDA_ERROR( hipMemcpy(rawDataIn_d[0], A, sizeA, hipMemcpyHostToDevice) );
   HANDLE_CUDA_ERROR( hipMemcpy(rawDataIn_d[1], B, sizeB, hipMemcpyHostToDevice) );
   HANDLE_CUDA_ERROR( hipMemcpy(rawDataIn_d[2], C, sizeC, hipMemcpyHostToDevice) );
   HANDLE_CUDA_ERROR( hipMemcpy(rawDataIn_d[3], D, sizeD, hipMemcpyHostToDevice) );

   if(verbose)
      printf("Allocated GPU memory for data, and initialize data\n");

   // Sphinx: #4
   /*************************
   * cuTensorNet
   *************************/

   hipStream_t stream;
   HANDLE_CUDA_ERROR( hipStreamCreate(&stream) );

   cutensornetHandle_t handle;
   HANDLE_ERROR( cutensornetCreate(&handle) );

   const int32_t nmodeA = modesA.size();
   const int32_t nmodeB = modesB.size();
   const int32_t nmodeC = modesC.size();
   const int32_t nmodeD = modesD.size();
   const int32_t nmodeR = modesR.size();

   /*******************************
   * Create Network Descriptor
   *******************************/

   const int32_t* modesIn[] = {modesA.data(), modesB.data(), modesC.data(), modesD.data()};
   int32_t const numModesIn[] = {nmodeA, nmodeB, nmodeC, nmodeD};
   const int64_t* extentsIn[] = {extentA.data(), extentB.data(), extentC.data(), extentD.data()};
   const int64_t* stridesIn[] = {NULL, NULL, NULL, NULL}; // strides are optional; if no stride is provided, cuTensorNet assumes a generalized column-major data layout

   // Set up tensor network
   cutensornetNetworkDescriptor_t descNet;
   HANDLE_ERROR( cutensornetCreateNetworkDescriptor(handle,
                     numInputs, numModesIn, extentsIn, stridesIn, modesIn, NULL,
                     nmodeR, extentR.data(), /*stridesOut = */NULL, modesR.data(),
                     typeData, typeCompute,
                     &descNet) );

   if(verbose)
      printf("Initialized the cuTensorNet library and created a tensor network descriptor\n");

   // Sphinx: #5
   /*******************************
   * Choose workspace limit based on available resources.
   *******************************/

   size_t freeMem, totalMem;
   HANDLE_CUDA_ERROR( hipMemGetInfo(&freeMem, &totalMem) );
   uint64_t workspaceLimit = (uint64_t)((double)freeMem * 0.9);
   if(verbose)
      printf("Workspace limit = %lu\n", workspaceLimit);

   /*******************************
   * Find "optimal" contraction order and slicing
   *******************************/

   cutensornetContractionOptimizerConfig_t optimizerConfig;
   HANDLE_ERROR( cutensornetCreateContractionOptimizerConfig(handle, &optimizerConfig) );

   // Set the desired number of hyper-samples (defaults to 0)
   int32_t num_hypersamples = 8;
   HANDLE_ERROR( cutensornetContractionOptimizerConfigSetAttribute(handle,
                     optimizerConfig,
                     CUTENSORNET_CONTRACTION_OPTIMIZER_CONFIG_HYPER_NUM_SAMPLES,
                     &num_hypersamples,
                     sizeof(num_hypersamples)) );

   // Create contraction optimizer info and find an optimized contraction path
   cutensornetContractionOptimizerInfo_t optimizerInfo;
   HANDLE_ERROR( cutensornetCreateContractionOptimizerInfo(handle, descNet, &optimizerInfo) );

   HANDLE_ERROR( cutensornetContractionOptimize(handle,
                                             descNet,
                                             optimizerConfig,
                                             workspaceLimit,
                                             optimizerInfo) );

   // Query the number of slices the tensor network execution will be split into
   int64_t numSlices = 0;
   HANDLE_ERROR( cutensornetContractionOptimizerInfoGetAttribute(
                  handle,
                  optimizerInfo,
                  CUTENSORNET_CONTRACTION_OPTIMIZER_INFO_NUM_SLICES,
                  &numSlices,
                  sizeof(numSlices)) );
   assert(numSlices > 0);

   if(verbose)
      printf("Found an optimized contraction path using cuTensorNet optimizer\n");

   // Sphinx: #6
   /*******************************
   * Create workspace descriptor, allocate workspace, and set it.
   *******************************/

   cutensornetWorkspaceDescriptor_t workDesc;
   HANDLE_ERROR( cutensornetCreateWorkspaceDescriptor(handle, &workDesc) );

   int64_t requiredWorkspaceSize = 0;
   HANDLE_ERROR( cutensornetWorkspaceComputeContractionSizes(handle,
                                                         descNet,
                                                         optimizerInfo,
                                                         workDesc) );

   HANDLE_ERROR( cutensornetWorkspaceGetMemorySize(handle,
                                                   workDesc,
                                                   CUTENSORNET_WORKSIZE_PREF_MIN,
                                                   CUTENSORNET_MEMSPACE_DEVICE,
                                                   CUTENSORNET_WORKSPACE_SCRATCH,
                                                   &requiredWorkspaceSize) );

   void* work = nullptr;
   HANDLE_CUDA_ERROR( hipMalloc(&work, requiredWorkspaceSize) );

   HANDLE_ERROR( cutensornetWorkspaceSetMemory(handle,
                                               workDesc,
                                               CUTENSORNET_MEMSPACE_DEVICE,
                                               CUTENSORNET_WORKSPACE_SCRATCH,
                                               work,
                                               requiredWorkspaceSize) );

   if(verbose)
      printf("Allocated and set up the GPU workspace\n");

   // Sphinx: #7
   /*******************************
   * Initialize the pairwise contraction plan (for cuTENSOR).
   *******************************/

   cutensornetContractionPlan_t plan;
   HANDLE_ERROR( cutensornetCreateContractionPlan(handle,
                                                descNet,
                                                optimizerInfo,
                                                workDesc,
                                                &plan) );

   /*******************************
   * Optional: Auto-tune cuTENSOR's cutensorContractionPlan to pick the fastest kernel
   *           for each pairwise tensor contraction.
   *******************************/
   cutensornetContractionAutotunePreference_t autotunePref;
   HANDLE_ERROR( cutensornetCreateContractionAutotunePreference(handle,
                                                      &autotunePref) );

   const int numAutotuningIterations = 5; // may be 0
   HANDLE_ERROR( cutensornetContractionAutotunePreferenceSetAttribute(
                           handle,
                           autotunePref,
                           CUTENSORNET_CONTRACTION_AUTOTUNE_MAX_ITERATIONS,
                           &numAutotuningIterations,
                           sizeof(numAutotuningIterations)) );

   // Modify the plan again to find the best pair-wise contractions
   HANDLE_ERROR( cutensornetContractionAutotune(handle,
                                                plan,
                                                rawDataIn_d,
                                                R_d,
                                                workDesc,
                                                autotunePref,
                                                stream) );

   HANDLE_ERROR( cutensornetDestroyContractionAutotunePreference(autotunePref) );

   if(verbose)
      printf("Created a contraction plan for cuTensorNet and optionally auto-tuned it\n");

   // Sphinx: #8
   /**********************
   * Execute the tensor network contraction
   **********************/

   // Create a cutensornetSliceGroup_t object from a range of slice IDs
   cutensornetSliceGroup_t sliceGroup{};
   HANDLE_ERROR( cutensornetCreateSliceGroupFromIDRange(handle, 0, numSlices, 1, &sliceGroup) );

   GPUTimer timer {stream};
   double minTimeCUTENSORNET = 1e100;
   const int numRuns = 3; // number of repeats to get stable performance results
   for (int i = 0; i < numRuns; ++i)
   {
      HANDLE_CUDA_ERROR( hipMemcpy(R_d, R, sizeR, hipMemcpyHostToDevice) ); // restore the output tensor on GPU
      HANDLE_CUDA_ERROR( hipDeviceSynchronize() );

      /*
      * Contract all slices of the tensor network
      */
      timer.start();

      int32_t accumulateOutput = 0; // output tensor data will be overwritten
      HANDLE_ERROR( cutensornetContractSlices(handle,
                     plan,
                     rawDataIn_d,
                     R_d,
                     accumulateOutput,
                     workDesc,
                     sliceGroup, // alternatively, NULL can also be used to contract over all slices instead of specifying a sliceGroup object
                     stream) );

      // Synchronize and measure best timing
      auto time = timer.seconds();
      minTimeCUTENSORNET = (time > minTimeCUTENSORNET) ? minTimeCUTENSORNET : time;
   }

   if(verbose)
      printf("Contracted the tensor network, each slice used the same contraction plan\n");

   // Print the 1-norm of the output tensor (verification)
   HANDLE_CUDA_ERROR( hipStreamSynchronize(stream) );
   HANDLE_CUDA_ERROR( hipMemcpy(R, R_d, sizeR, hipMemcpyDeviceToHost) ); // restore the output tensor on Host
   double norm1 = 0.0;
   for (int64_t i = 0; i < elementsR; ++i) {
      norm1 += std::abs(R[i]);
   }
   if(verbose)
      printf("Computed the 1-norm of the output tensor: %e\n", norm1);

   /*************************/

   // Query the total Flop count for the tensor network contraction
   double flops {0.0};
   HANDLE_ERROR( cutensornetContractionOptimizerInfoGetAttribute(
                     handle,
                     optimizerInfo,
                     CUTENSORNET_CONTRACTION_OPTIMIZER_INFO_FLOP_COUNT,
                     &flops,
                     sizeof(flops)) );

   if(verbose) {
      printf("Number of tensor network slices = %ld\n", numSlices);
      printf("Tensor network contraction time (ms) = %.3f\n", minTimeCUTENSORNET * 1000.f);
   }

   // Free cuTensorNet resources
   HANDLE_ERROR( cutensornetDestroySliceGroup(sliceGroup) );
   HANDLE_ERROR( cutensornetDestroyContractionPlan(plan) );
   HANDLE_ERROR( cutensornetDestroyWorkspaceDescriptor(workDesc) );
   HANDLE_ERROR( cutensornetDestroyContractionOptimizerInfo(optimizerInfo) );
   HANDLE_ERROR( cutensornetDestroyContractionOptimizerConfig(optimizerConfig) );
   HANDLE_ERROR( cutensornetDestroyNetworkDescriptor(descNet) );
   HANDLE_ERROR( cutensornetDestroy(handle) );

   // Free Host memory resources
   if (R) free(R);
   if (D) free(D);
   if (C) free(C);
   if (B) free(B);
   if (A) free(A);

   // Free GPU memory resources
   if (work) hipFree(work);
   if (R_d) hipFree(R_d);
   if (rawDataIn_d[0]) hipFree(rawDataIn_d[0]);
   if (rawDataIn_d[1]) hipFree(rawDataIn_d[1]);
   if (rawDataIn_d[2]) hipFree(rawDataIn_d[2]);
   if (rawDataIn_d[3]) hipFree(rawDataIn_d[3]);

   if(verbose)
      printf("Freed resources and exited\n");

   return 0;
}
