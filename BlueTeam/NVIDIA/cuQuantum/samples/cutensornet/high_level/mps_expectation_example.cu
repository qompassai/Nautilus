#include "hip/hip_runtime.h"
/* Copyright (c) 2023-2024, NVIDIA CORPORATION & AFFILIATES.
 *
 * SPDX-License-Identifier: BSD-3-Clause
 */

// Sphinx: MPS Expectation #1

#include <cstdlib>
#include <cstdio>
#include <cassert>
#include <complex>
#include <vector>
#include <bitset>
#include <iostream>

#include <hip/hip_runtime.h>
#include <cutensornet.h>


#define HANDLE_CUDA_ERROR(x) \
{ const auto err = x; \
  if( err != hipSuccess ) \
  { printf("CUDA error %s in line %d\n", hipGetErrorString(err), __LINE__); fflush(stdout); std::abort(); } \
};

#define HANDLE_CUTN_ERROR(x) \
{ const auto err = x; \
  if( err != CUTENSORNET_STATUS_SUCCESS ) \
  { printf("cuTensorNet error %s in line %d\n", cutensornetGetErrorString(err), __LINE__); fflush(stdout); std::abort(); } \
};


int main()
{
  static_assert(sizeof(size_t) == sizeof(int64_t), "Please build this sample on a 64-bit architecture!");

  constexpr std::size_t fp64size = sizeof(double);

  // Sphinx: MPS Expectation #2

  // Quantum state configuration
  constexpr int32_t numQubits = 16; // number of qubits
  const std::vector<int64_t> qubitDims(numQubits,2); // qubit dimensions
  std::cout << "Quantum circuit: " << numQubits << " qubits\n";

  // Sphinx: MPS Expectation #3

  // Initialize the cuTensorNet library
  HANDLE_CUDA_ERROR(hipSetDevice(0));
  cutensornetHandle_t cutnHandle;
  HANDLE_CUTN_ERROR(cutensornetCreate(&cutnHandle));
  std::cout << "Initialized cuTensorNet library on GPU 0\n";

  // Sphinx: MPS Expectation #4

  // Define necessary quantum gate tensors in Host memory
  const double invsq2 = 1.0 / std::sqrt(2.0);
  //  Hadamard gate
  const std::vector<std::complex<double>> h_gateH {{invsq2, 0.0},  {invsq2, 0.0},
                                                   {invsq2, 0.0}, {-invsq2, 0.0}};
  //  Pauli X gate
  const std::vector<std::complex<double>> h_gateX {{0.0, 0.0}, {1.0, 0.0},
                                                   {1.0, 0.0}, {0.0, 0.0}};
  //  Pauli Y gate
  const std::vector<std::complex<double>> h_gateY {{0.0, 0.0}, {0.0, -1.0},
                                                   {0.0, 1.0}, {0.0, 0.0}};
  //  Pauli Z gate
  const std::vector<std::complex<double>> h_gateZ {{1.0, 0.0}, {0.0, 0.0},
                                                   {0.0, 0.0}, {-1.0, 0.0}};
  //  CX gate
  const std::vector<std::complex<double>> h_gateCX {{1.0, 0.0}, {0.0, 0.0}, {0.0, 0.0}, {0.0, 0.0},
                                                    {0.0, 0.0}, {1.0, 0.0}, {0.0, 0.0}, {0.0, 0.0},
                                                    {0.0, 0.0}, {0.0, 0.0}, {0.0, 0.0}, {1.0, 0.0},
                                                    {0.0, 0.0}, {0.0, 0.0}, {1.0, 0.0}, {0.0, 0.0}};

  // Copy quantum gates to Device memory
  void *d_gateH{nullptr}, *d_gateX{nullptr}, *d_gateY{nullptr}, *d_gateZ{nullptr}, *d_gateCX{nullptr};
  HANDLE_CUDA_ERROR(hipMalloc(&d_gateH, 4 * (2 * fp64size)));
  HANDLE_CUDA_ERROR(hipMalloc(&d_gateX, 4 * (2 * fp64size)));
  HANDLE_CUDA_ERROR(hipMalloc(&d_gateY, 4 * (2 * fp64size)));
  HANDLE_CUDA_ERROR(hipMalloc(&d_gateZ, 4 * (2 * fp64size)));
  HANDLE_CUDA_ERROR(hipMalloc(&d_gateCX, 16 * (2 * fp64size)));
  std::cout << "Allocated quantum gate memory on GPU\n";
  HANDLE_CUDA_ERROR(hipMemcpy(d_gateH, h_gateH.data(), 4 * (2 * fp64size), hipMemcpyHostToDevice));
  HANDLE_CUDA_ERROR(hipMemcpy(d_gateX, h_gateX.data(), 4 * (2 * fp64size), hipMemcpyHostToDevice));
  HANDLE_CUDA_ERROR(hipMemcpy(d_gateY, h_gateY.data(), 4 * (2 * fp64size), hipMemcpyHostToDevice));
  HANDLE_CUDA_ERROR(hipMemcpy(d_gateZ, h_gateZ.data(), 4 * (2 * fp64size), hipMemcpyHostToDevice));
  HANDLE_CUDA_ERROR(hipMemcpy(d_gateCX, h_gateCX.data(), 16 * (2 * fp64size), hipMemcpyHostToDevice));
  std::cout << "Copied quantum gates to GPU memory\n";

  // Sphinx: MPS Expectation #5

  // Determine the MPS representation and allocate buffers for the MPS tensors
  const int64_t maxExtent = 2; // GHZ state can be exactly represented with max bond dimension of 2
  std::vector<std::vector<int64_t>> extents;
  std::vector<int64_t*> extentsPtr(numQubits); 
  std::vector<void*> d_mpsTensors(numQubits, nullptr);
  for (int32_t i = 0; i < numQubits; i++) {
    if (i == 0) { // left boundary MPS tensor
      extents.push_back({2, maxExtent});
      HANDLE_CUDA_ERROR(hipMalloc(&d_mpsTensors[i], 2 * maxExtent * 2 * fp64size));
    } 
    else if (i == numQubits-1) { // right boundary MPS tensor
      extents.push_back({maxExtent, 2});
      HANDLE_CUDA_ERROR(hipMalloc(&d_mpsTensors[i], 2 * maxExtent * 2 * fp64size));
    }
    else { // middle MPS tensors
      extents.push_back({maxExtent, 2, maxExtent});
      HANDLE_CUDA_ERROR(hipMalloc(&d_mpsTensors[i], 2 * maxExtent * maxExtent * 2 * fp64size));
    }
    extentsPtr[i] = extents[i].data();
  }

  // Sphinx: MPS Expectation #6

  // Query the free memory on Device
  std::size_t freeSize{0}, totalSize{0};
  HANDLE_CUDA_ERROR(hipMemGetInfo(&freeSize, &totalSize));
  const std::size_t scratchSize = (freeSize - (freeSize % 4096)) / 2; // use half of available memory with alignment
  void *d_scratch{nullptr};
  HANDLE_CUDA_ERROR(hipMalloc(&d_scratch, scratchSize));
  std::cout << "Allocated " << scratchSize << " bytes of scratch memory on GPU\n";

  // Sphinx: MPS Expectation #7

  // Create the initial quantum state
  cutensornetState_t quantumState;
  HANDLE_CUTN_ERROR(cutensornetCreateState(cutnHandle, CUTENSORNET_STATE_PURITY_PURE, numQubits, qubitDims.data(),
                    HIP_C_64F, &quantumState));
  std::cout << "Created the initial quantum state\n";

  // Sphinx: MPS Expectation #8

  // Construct the final quantum circuit state (apply quantum gates) for the GHZ circuit
  int64_t id;
  HANDLE_CUTN_ERROR(cutensornetStateApplyTensorOperator(cutnHandle, quantumState, 1, std::vector<int32_t>{{0}}.data(),
                    d_gateH, nullptr, 1, 0, 1, &id));
  for(int32_t i = 1; i < numQubits; ++i) {
    HANDLE_CUTN_ERROR(cutensornetStateApplyTensorOperator(cutnHandle, quantumState, 2, std::vector<int32_t>{{i-1,i}}.data(),
                      d_gateCX, nullptr, 1, 0, 1, &id));
  }
  std::cout << "Applied quantum gates\n";

  // Sphinx: MPS Expectation #9

  // Specify the final target MPS representation (use default fortran strides)
  HANDLE_CUTN_ERROR(cutensornetStateFinalizeMPS(cutnHandle, quantumState, 
                    CUTENSORNET_BOUNDARY_CONDITION_OPEN, extentsPtr.data(), /*strides=*/nullptr ));

  // Sphinx: MPS Expectation #10

  // Optional, set up the SVD method for truncation.
  cutensornetTensorSVDAlgo_t algo = CUTENSORNET_TENSOR_SVD_ALGO_GESVDJ; 
  HANDLE_CUTN_ERROR(cutensornetStateConfigure(cutnHandle, quantumState, 
                    CUTENSORNET_STATE_CONFIG_MPS_SVD_ALGO, &algo, sizeof(algo)));
  std::cout << "Configured the MPS computation\n";

  // Sphinx: MPS Expectation #11

  // Prepare the MPS computation and attach workspace
  cutensornetWorkspaceDescriptor_t workDesc;
  HANDLE_CUTN_ERROR(cutensornetCreateWorkspaceDescriptor(cutnHandle, &workDesc));
  std::cout << "Created the workspace descriptor\n";
  HANDLE_CUTN_ERROR(cutensornetStatePrepare(cutnHandle, quantumState, scratchSize, workDesc, 0x0));
  std::cout << "Prepared the computation of the quantum circuit state\n";
  double flops {0.0};
  HANDLE_CUTN_ERROR(cutensornetStateGetInfo(cutnHandle, quantumState,
                    CUTENSORNET_STATE_INFO_FLOPS, &flops, sizeof(flops)));
  if(flops > 0.0) {
    std::cout << "Total flop count = " << (flops/1e9) << " GFlop\n";
  }else if(flops < 0.0) {
    std::cout << "ERROR: Negative Flop count!\n";
    std::abort();
  }

  int64_t worksize {0};
  HANDLE_CUTN_ERROR(cutensornetWorkspaceGetMemorySize(cutnHandle,
                                                      workDesc,
                                                      CUTENSORNET_WORKSIZE_PREF_RECOMMENDED,
                                                      CUTENSORNET_MEMSPACE_DEVICE,
                                                      CUTENSORNET_WORKSPACE_SCRATCH,
                                                      &worksize));
  std::cout << "Scratch GPU workspace size (bytes) for MPS computation = " << worksize << std::endl;
  if(worksize <= scratchSize) {
    HANDLE_CUTN_ERROR(cutensornetWorkspaceSetMemory(cutnHandle, workDesc, CUTENSORNET_MEMSPACE_DEVICE,
                      CUTENSORNET_WORKSPACE_SCRATCH, d_scratch, worksize));
  }else{
    std::cout << "ERROR: Insufficient workspace size on Device!\n";
    std::abort();
  }
  std::cout << "Set the workspace buffer for MPS computation\n";

  // Sphinx: MPS Expectation #12

  // Execute MPS computation
  HANDLE_CUTN_ERROR(cutensornetStateCompute(cutnHandle, quantumState, 
                    workDesc, extentsPtr.data(), /*strides=*/nullptr, d_mpsTensors.data(), 0));

  // Sphinx: MPS Expectation #13

  // Create an empty tensor network operator
  cutensornetNetworkOperator_t hamiltonian;
  HANDLE_CUTN_ERROR(cutensornetCreateNetworkOperator(cutnHandle, numQubits, qubitDims.data(), HIP_C_64F, &hamiltonian));
  // Append component (0.5 * Z1 * Z2) to the tensor network operator
  {
    const int32_t numModes[] = {1, 1}; // Z1 acts on 1 mode, Z2 acts on 1 mode
    const int32_t modesZ1[] = {1}; // state modes Z1 acts on
    const int32_t modesZ2[] = {2}; // state modes Z2 acts on
    const int32_t * stateModes[] = {modesZ1, modesZ2}; // state modes (Z1 * Z2) acts on
    const void * gateData[] = {d_gateZ, d_gateZ}; // GPU pointers to gate data
    HANDLE_CUTN_ERROR(cutensornetNetworkOperatorAppendProduct(cutnHandle, hamiltonian, hipDoubleComplex{0.5,0.0},
                      2, numModes, stateModes, NULL, gateData, &id));
  }
  // Append component (0.25 * Y3) to the tensor network operator
  {
    const int32_t numModes[] = {1}; // Y3 acts on 1 mode
    const int32_t modesY3[] = {3}; // state modes Y3 acts on
    const int32_t * stateModes[] = {modesY3}; // state modes (Y3) acts on
    const void * gateData[] = {d_gateY}; // GPU pointers to gate data
    HANDLE_CUTN_ERROR(cutensornetNetworkOperatorAppendProduct(cutnHandle, hamiltonian, hipDoubleComplex{0.25,0.0},
                      1, numModes, stateModes, NULL, gateData, &id));
  }
  // Append component (0.13 * Y0 X2 Z3) to the tensor network operator
  {
    const int32_t numModes[] = {1, 1, 1}; // Y0 acts on 1 mode, X2 acts on 1 mode, Z3 acts on 1 mode
    const int32_t modesY0[] = {0}; // state modes Y0 acts on
    const int32_t modesX2[] = {2}; // state modes X2 acts on
    const int32_t modesZ3[] = {3}; // state modes Z3 acts on
    const int32_t * stateModes[] = {modesY0, modesX2, modesZ3}; // state modes (Y0 * X2 * Z3) acts on
    const void * gateData[] = {d_gateY, d_gateX, d_gateZ}; // GPU pointers to gate data
    HANDLE_CUTN_ERROR(cutensornetNetworkOperatorAppendProduct(cutnHandle, hamiltonian, hipDoubleComplex{0.13,0.0},
                      3, numModes, stateModes, NULL, gateData, &id));
  }
  std::cout << "Constructed a tensor network operator: (0.5 * Z1 * Z2) + (0.25 * Y3) + (0.13 * Y0 * X2 * Z3)" << std::endl;

  // Sphinx: MPS Expectation #14

  // Specify the quantum circuit expectation value
  cutensornetStateExpectation_t expectation;
  HANDLE_CUTN_ERROR(cutensornetCreateExpectation(cutnHandle, quantumState, hamiltonian, &expectation));
  std::cout << "Created the specified quantum circuit expectation value\n";

  // Sphinx: MPS Expectation #15

  // Configure the computation of the specified quantum circuit expectation value
  const int32_t numHyperSamples = 8; // desired number of hyper samples used in the tensor network contraction path finder
  HANDLE_CUTN_ERROR(cutensornetExpectationConfigure(cutnHandle, expectation,
                    CUTENSORNET_EXPECTATION_CONFIG_NUM_HYPER_SAMPLES, &numHyperSamples, sizeof(numHyperSamples)));

  // Sphinx: MPS Expectation #16

  // Prepare the specified quantum circuit expectation value for computation
  HANDLE_CUTN_ERROR(cutensornetExpectationPrepare(cutnHandle, expectation, scratchSize, workDesc, 0x0));
  std::cout << "Prepared the specified quantum circuit expectation value\n";
  flops = 0.0;
  HANDLE_CUTN_ERROR(cutensornetExpectationGetInfo(cutnHandle, expectation,
                    CUTENSORNET_EXPECTATION_INFO_FLOPS, &flops, sizeof(flops)));
  std::cout << "Total flop count = " << (flops/1e9) << " GFlop\n";
  if(flops <= 0.0) {
    std::cout << "ERROR: Invalid Flop count!\n";
    std::abort();
  }

  // Sphinx: MPS Expectation #17

  // Attach the workspace buffer
  HANDLE_CUTN_ERROR(cutensornetWorkspaceGetMemorySize(cutnHandle,
                                                      workDesc,
                                                      CUTENSORNET_WORKSIZE_PREF_RECOMMENDED,
                                                      CUTENSORNET_MEMSPACE_DEVICE,
                                                      CUTENSORNET_WORKSPACE_SCRATCH,
                                                      &worksize));
  std::cout << "Required scratch GPU workspace size (bytes) = " << worksize << std::endl;
  if(worksize <= scratchSize) {
    HANDLE_CUTN_ERROR(cutensornetWorkspaceSetMemory(cutnHandle, workDesc, CUTENSORNET_MEMSPACE_DEVICE,
                      CUTENSORNET_WORKSPACE_SCRATCH, d_scratch, worksize));
  }else{
    std::cout << "ERROR: Insufficient workspace size on Device!\n";
    std::abort();
  }
  std::cout << "Set the workspace buffer\n";

  // Sphinx: MPS Expectation #18

  // Compute the specified quantum circuit expectation value
  std::complex<double> expectVal{0.0,0.0}, stateNorm2{0.0,0.0};
  HANDLE_CUTN_ERROR(cutensornetExpectationCompute(cutnHandle, expectation, workDesc,
                    static_cast<void*>(&expectVal), static_cast<void*>(&stateNorm2), 0x0));
  std::cout << "Computed the specified quantum circuit expectation value\n";
  expectVal /= stateNorm2;
  std::cout << "Expectation value = (" << expectVal.real() << ", " << expectVal.imag() << ")\n";
  std::cout << "Squared 2-norm of the state = (" << stateNorm2.real() << ", " << stateNorm2.imag() << ")\n";

  // Sphinx: MPS Expectation #19

  // Destroy the workspace descriptor
  HANDLE_CUTN_ERROR(cutensornetDestroyWorkspaceDescriptor(workDesc));
  std::cout << "Destroyed the workspace descriptor\n";

  // Destroy the quantum circuit expectation value
  HANDLE_CUTN_ERROR(cutensornetDestroyExpectation(expectation));
  std::cout << "Destroyed the quantum circuit state expectation value\n";

  // Destroy the tensor network operator
  HANDLE_CUTN_ERROR(cutensornetDestroyNetworkOperator(hamiltonian));
  std::cout << "Destroyed the tensor network operator\n";

  // Destroy the quantum circuit state
  HANDLE_CUTN_ERROR(cutensornetDestroyState(quantumState));
  std::cout << "Destroyed the quantum circuit state\n";

  for (int32_t i = 0; i < numQubits; i++) {
    HANDLE_CUDA_ERROR(hipFree(d_mpsTensors[i]));
  }
  HANDLE_CUDA_ERROR(hipFree(d_scratch));
  HANDLE_CUDA_ERROR(hipFree(d_gateCX));
  HANDLE_CUDA_ERROR(hipFree(d_gateZ));
  HANDLE_CUDA_ERROR(hipFree(d_gateY));
  HANDLE_CUDA_ERROR(hipFree(d_gateX));
  HANDLE_CUDA_ERROR(hipFree(d_gateH));
  std::cout << "Freed memory on GPU\n";

  // Finalize the cuTensorNet library
  HANDLE_CUTN_ERROR(cutensornetDestroy(cutnHandle));
  std::cout << "Finalized the cuTensorNet library\n";

  return 0;
}
