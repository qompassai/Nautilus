#include "hip/hip_runtime.h"
/* Copyright (c) 2024, NVIDIA CORPORATION & AFFILIATES.
 *
 * SPDX-License-Identifier: BSD-3-Clause
 */

// Sphinx: Amplitudes #1

#include <cstdlib>
#include <cstdio>
#include <cassert>
#include <complex>
#include <vector>
#include <bitset>
#include <iostream>

#include <hip/hip_runtime.h>
#include <cutensornet.h>


#define HANDLE_CUDA_ERROR(x) \
{ const auto err = x; \
  if( err != hipSuccess ) \
  { printf("CUDA error %s in line %d\n", hipGetErrorString(err), __LINE__); fflush(stdout); std::abort(); } \
};

#define HANDLE_CUTN_ERROR(x) \
{ const auto err = x; \
  if( err != CUTENSORNET_STATUS_SUCCESS ) \
  { printf("cuTensorNet error %s in line %d\n", cutensornetGetErrorString(err), __LINE__); fflush(stdout); std::abort(); } \
};


int main()
{
  static_assert(sizeof(size_t) == sizeof(int64_t), "Please build this sample on a 64-bit architecture!");

  constexpr std::size_t fp64size = sizeof(double);

  // Sphinx: Amplitudes #2

  // Quantum state configuration
  constexpr int32_t numQubits = 6; // number of qubits
  const std::vector<int64_t> qubitDims(numQubits,2); // qubit dimensions
  const std::vector<int32_t> fixedModes({0,1}); // fixed modes in the output amplitude tensor (must be in acsending order)
  const std::vector<int64_t> fixedValues({1,1}); // values of the fixed modes in the output amplitude tensor
  const int32_t numFixedModes = fixedModes.size(); // number of fixed modes in the output amplitude tensor
  std::cout << "Quantum circuit: " << numQubits << " qubits\n";

  // Sphinx: Amplitudes #3

  // Initialize the cuTensorNet library
  HANDLE_CUDA_ERROR(hipSetDevice(0));
  cutensornetHandle_t cutnHandle;
  HANDLE_CUTN_ERROR(cutensornetCreate(&cutnHandle));
  std::cout << "Initialized cuTensorNet library on GPU 0\n";

  // Sphinx: Amplitudes #4

  // Define necessary quantum gate tensors in Host memory
  const double invsq2 = 1.0 / std::sqrt(2.0);
  //  Hadamard gate
  const std::vector<std::complex<double>> h_gateH {{invsq2, 0.0},  {invsq2, 0.0},
                                                   {invsq2, 0.0}, {-invsq2, 0.0}};
  //  X gate
  const std::vector<std::complex<double>> h_gateX {{0.0, 0.0}, {1.0, 0.0},
                                                    {1.0, 0.0}, {0.0, 0.0}};

  // Copy quantum gates to Device memory
  void *d_gateH{nullptr}, *d_gateX{nullptr};
  HANDLE_CUDA_ERROR(hipMalloc(&d_gateH, 4 * (2 * fp64size)));
  HANDLE_CUDA_ERROR(hipMalloc(&d_gateX, 4 * (2 * fp64size)));
  std::cout << "Allocated quantum gate memory on GPU\n";
  HANDLE_CUDA_ERROR(hipMemcpy(d_gateH, h_gateH.data(), 4 * (2 * fp64size), hipMemcpyHostToDevice));
  HANDLE_CUDA_ERROR(hipMemcpy(d_gateX, h_gateX.data(), 4 * (2 * fp64size), hipMemcpyHostToDevice));
  std::cout << "Copied quantum gates to GPU memory\n";

  // Sphinx: Amplitudes #5

  // Allocate Device memory for the specified slice of the quantum circuit amplitudes tensor
  void *d_amp{nullptr};
  std::size_t ampSize = 1;
  for(const auto & qubitDim: qubitDims) ampSize *= qubitDim; // all state modes (full size)
  for(const auto & fixedMode: fixedModes) ampSize /= qubitDims[fixedMode]; // fixed state modes reduce the slice size
  HANDLE_CUDA_ERROR(hipMalloc(&d_amp, ampSize * (2 * fp64size)));
  std::cout << "Allocated memory for the specified slice of the quantum circuit amplitude tensor of size "
            << ampSize << " elements\n";

  // Sphinx: Amplitudes #6

  // Query the free memory on Device
  std::size_t freeSize{0}, totalSize{0};
  HANDLE_CUDA_ERROR(hipMemGetInfo(&freeSize, &totalSize));
  const std::size_t scratchSize = (freeSize - (freeSize % 4096)) / 2; // use half of available memory with alignment
  void *d_scratch{nullptr};
  HANDLE_CUDA_ERROR(hipMalloc(&d_scratch, scratchSize));
  std::cout << "Allocated " << scratchSize << " bytes of scratch memory on GPU\n";

  // Sphinx: Amplitudes #7

  // Create the initial quantum state
  cutensornetState_t quantumState;
  HANDLE_CUTN_ERROR(cutensornetCreateState(cutnHandle, CUTENSORNET_STATE_PURITY_PURE, numQubits, qubitDims.data(),
                    HIP_C_64F, &quantumState));
  std::cout << "Created the initial quantum state\n";

  // Sphinx: Amplitudes #8

  // Construct the final quantum circuit state (apply quantum gates) for the GHZ circuit
  int64_t id;
  HANDLE_CUTN_ERROR(cutensornetStateApplyTensorOperator(cutnHandle, quantumState, 1, std::vector<int32_t>{{0}}.data(),
                    d_gateH, nullptr, 1, 0, 1, &id));
  for(int32_t i = 1; i < numQubits; ++i) {
    // Apply Controlled-X gates
    HANDLE_CUTN_ERROR( cutensornetStateApplyControlledTensorOperator(cutnHandle, quantumState, 1, std::vector<int32_t>{{i-1}}.data(),
    nullptr , 1, std::vector<int32_t>{{i}}.data(), d_gateX, nullptr, 1, 0, 1, &id  ) );
  }
  std::cout << "Applied quantum gates\n";

  // Sphinx: Amplitudes #9

  // Specify the quantum circuit amplitudes accessor
  cutensornetStateAccessor_t accessor;
  HANDLE_CUTN_ERROR(cutensornetCreateAccessor(cutnHandle, quantumState, numFixedModes, fixedModes.data(),
                    nullptr, &accessor)); // using default strides
  std::cout << "Created the specified quantum circuit amplitudes accessor\n";

  // Sphinx: Amplitudes #10

  // Configure the computation of the slice of the specified quantum circuit amplitudes tensor
  const int32_t numHyperSamples = 8; // desired number of hyper samples used in the tensor network contraction path finder
  HANDLE_CUTN_ERROR(cutensornetAccessorConfigure(cutnHandle, accessor,
                    CUTENSORNET_ACCESSOR_CONFIG_NUM_HYPER_SAMPLES, &numHyperSamples, sizeof(numHyperSamples)));

  // Sphinx: Amplitudes #11

  // Prepare the computation of the specified slice of the quantum circuit amplitudes tensor
  cutensornetWorkspaceDescriptor_t workDesc;
  HANDLE_CUTN_ERROR(cutensornetCreateWorkspaceDescriptor(cutnHandle, &workDesc));
  std::cout << "Created the workspace descriptor\n";
  HANDLE_CUTN_ERROR(cutensornetAccessorPrepare(cutnHandle, accessor, scratchSize, workDesc, 0x0));
  std::cout << "Prepared the computation of the specified slice of the quantum circuit amplitudes tensor\n";
  double flops {0.0};
  HANDLE_CUTN_ERROR(cutensornetAccessorGetInfo(cutnHandle, accessor,
                    CUTENSORNET_ACCESSOR_INFO_FLOPS, &flops, sizeof(flops)));
  std::cout << "Total flop count = " << (flops/1e9) << " GFlop\n";

  // Sphinx: Amplitudes #12

  // Attach the workspace buffer
  int64_t worksize {0};
  HANDLE_CUTN_ERROR(cutensornetWorkspaceGetMemorySize(cutnHandle,
                                                      workDesc,
                                                      CUTENSORNET_WORKSIZE_PREF_RECOMMENDED,
                                                      CUTENSORNET_MEMSPACE_DEVICE,
                                                      CUTENSORNET_WORKSPACE_SCRATCH,
                                                      &worksize));
  std::cout << "Required scratch GPU workspace size (bytes) = " << worksize << std::endl;
  if(worksize <= scratchSize) {
    HANDLE_CUTN_ERROR(cutensornetWorkspaceSetMemory(cutnHandle, workDesc, CUTENSORNET_MEMSPACE_DEVICE,
                      CUTENSORNET_WORKSPACE_SCRATCH, d_scratch, worksize));
  }else{
    std::cout << "ERROR: Insufficient workspace size on Device!\n";
    std::abort();
  }
  std::cout << "Set the workspace buffer\n";

  // Sphinx: Amplitudes #13

  // Compute the specified slice of the quantum circuit amplitudes tensor
  std::complex<double> stateNorm2{0.0,0.0};
  HANDLE_CUTN_ERROR(cutensornetAccessorCompute(cutnHandle, accessor, fixedValues.data(),
                    workDesc, d_amp, static_cast<void*>(&stateNorm2), 0x0));
  std::cout << "Computed the specified slice of the quantum circuit amplitudes tensor\n";
  std::vector<std::complex<double>> h_amp(ampSize);
  HANDLE_CUDA_ERROR(hipMemcpy(h_amp.data(), d_amp, ampSize * (2 * fp64size), hipMemcpyDeviceToHost));
  std::cout << "Amplitudes slice for " << (numQubits - numFixedModes) << " qubits:\n";
  for(std::size_t i = 0; i < ampSize; ++i) {
    std::cout << " " << h_amp[i] << std::endl;
  }
  std::cout << "Squared 2-norm of the state = (" << stateNorm2.real() << ", " << stateNorm2.imag() << ")\n";

  // Sphinx: Amplitudes #14

  // Destroy the workspace descriptor
  HANDLE_CUTN_ERROR(cutensornetDestroyWorkspaceDescriptor(workDesc));
  std::cout << "Destroyed the workspace descriptor\n";

  // Destroy the quantum circuit amplitudes accessor
  HANDLE_CUTN_ERROR(cutensornetDestroyAccessor(accessor));
  std::cout << "Destroyed the quantum circuit amplitudes accessor\n";

  // Destroy the quantum circuit state
  HANDLE_CUTN_ERROR(cutensornetDestroyState(quantumState));
  std::cout << "Destroyed the quantum circuit state\n";

  HANDLE_CUDA_ERROR(hipFree(d_scratch));
  HANDLE_CUDA_ERROR(hipFree(d_amp));
  HANDLE_CUDA_ERROR(hipFree(d_gateX));
  HANDLE_CUDA_ERROR(hipFree(d_gateH));
  std::cout << "Freed memory on GPU\n";

  // Finalize the cuTensorNet library
  HANDLE_CUTN_ERROR(cutensornetDestroy(cutnHandle));
  std::cout << "Finalized the cuTensorNet library\n";

  return 0;
}
