#include "hip/hip_runtime.h"
/* Copyright (c) 2023-2024, NVIDIA CORPORATION & AFFILIATES.
 *
 * SPDX-License-Identifier: BSD-3-Clause
 */

// Sphinx: MPS Marginal #1

#include <cstdlib>
#include <cstdio>
#include <cassert>
#include <complex>
#include <vector>
#include <bitset>
#include <iostream>

#include <hip/hip_runtime.h>
#include <cutensornet.h>


#define HANDLE_CUDA_ERROR(x) \
{ const auto err = x; \
  if( err != hipSuccess ) \
  { printf("CUDA error %s in line %d\n", hipGetErrorString(err), __LINE__); fflush(stdout); std::abort(); } \
};

#define HANDLE_CUTN_ERROR(x) \
{ const auto err = x; \
  if( err != CUTENSORNET_STATUS_SUCCESS ) \
  { printf("cuTensorNet error %s in line %d\n", cutensornetGetErrorString(err), __LINE__); fflush(stdout); std::abort(); } \
};


int main()
{
  static_assert(sizeof(size_t) == sizeof(int64_t), "Please build this sample on a 64-bit architecture!");

  constexpr std::size_t fp64size = sizeof(double);

  // Sphinx: MPS Marginal #2

  // Quantum state configuration
  constexpr int32_t numQubits = 16;
  const std::vector<int64_t> qubitDims(numQubits,2); // qubit dimensions
  constexpr int32_t numMarginalModes = 2; // rank of the marginal (reduced density matrix)
  const std::vector<int32_t> marginalModes({0,1}); // open qubits (must be in acsending order)
  std::cout << "Quantum circuit: " << numQubits << " qubits\n";

  // Sphinx: MPS Marginal #3

  // Initialize the cuTensorNet library
  HANDLE_CUDA_ERROR(hipSetDevice(0));
  cutensornetHandle_t cutnHandle;
  HANDLE_CUTN_ERROR(cutensornetCreate(&cutnHandle));
  std::cout << "Initialized cuTensorNet library on GPU 0\n";

  // Sphinx: MPS Marginal #4

  // Define necessary quantum gate tensors in Host memory
  const double invsq2 = 1.0 / std::sqrt(2.0);
  //  Hadamard gate
  const std::vector<std::complex<double>> h_gateH {{invsq2, 0.0},  {invsq2, 0.0},
                                                   {invsq2, 0.0}, {-invsq2, 0.0}};
  //  CX gate
  const std::vector<std::complex<double>> h_gateCX {{1.0, 0.0}, {0.0, 0.0}, {0.0, 0.0}, {0.0, 0.0},
                                                    {0.0, 0.0}, {1.0, 0.0}, {0.0, 0.0}, {0.0, 0.0},
                                                    {0.0, 0.0}, {0.0, 0.0}, {0.0, 0.0}, {1.0, 0.0},
                                                    {0.0, 0.0}, {0.0, 0.0}, {1.0, 0.0}, {0.0, 0.0}};

  // Copy quantum gates to Device memory
  void *d_gateH{nullptr}, *d_gateCX{nullptr};
  HANDLE_CUDA_ERROR(hipMalloc(&d_gateH, 4 * (2 * fp64size)));
  HANDLE_CUDA_ERROR(hipMalloc(&d_gateCX, 16 * (2 * fp64size)));
  std::cout << "Allocated quantum gate memory on GPU\n";
  HANDLE_CUDA_ERROR(hipMemcpy(d_gateH, h_gateH.data(), 4 * (2 * fp64size), hipMemcpyHostToDevice));
  HANDLE_CUDA_ERROR(hipMemcpy(d_gateCX, h_gateCX.data(), 16 * (2 * fp64size), hipMemcpyHostToDevice));
  std::cout << "Copied quantum gates to GPU memory\n";

  // Sphinx: MPS Marginal #5

  // Determine the MPS representation and allocate buffers for the MPS tensors
  const int64_t maxExtent = 2; // GHZ state can be exactly represented with max bond dimension of 2
  std::vector<std::vector<int64_t>> extents;
  std::vector<int64_t*> extentsPtr(numQubits); 
  std::vector<void*> d_mpsTensors(numQubits, nullptr);
  for (int32_t i = 0; i < numQubits; i++) {
    if (i == 0) { // left boundary MPS tensor
      extents.push_back({2, maxExtent});
      HANDLE_CUDA_ERROR(hipMalloc(&d_mpsTensors[i], 2 * maxExtent * 2 * fp64size));
    } 
    else if (i == numQubits-1) { // right boundary MPS tensor
      extents.push_back({maxExtent, 2});
      HANDLE_CUDA_ERROR(hipMalloc(&d_mpsTensors[i], 2 * maxExtent * 2 * fp64size));
    }
    else { // middle MPS tensors
      extents.push_back({maxExtent, 2, maxExtent});
      HANDLE_CUDA_ERROR(hipMalloc(&d_mpsTensors[i], 2 * maxExtent * maxExtent * 2 * fp64size));
    }
    extentsPtr[i] = extents[i].data();
  }

  // Sphinx: MPS Marginal #6

  // Allocate the specified quantum circuit reduced density matrix (marginal) in Device memory
  void *d_rdm{nullptr};
  std::size_t rdmDim = 1;
  for(const auto & mode: marginalModes) rdmDim *= qubitDims[mode];
  const std::size_t rdmSize = rdmDim * rdmDim;
  HANDLE_CUDA_ERROR(hipMalloc(&d_rdm, rdmSize * (2 * fp64size)));

  // Sphinx: MPS Marginal #7

  // Query the free memory on Device
  std::size_t freeSize{0}, totalSize{0};
  HANDLE_CUDA_ERROR(hipMemGetInfo(&freeSize, &totalSize));
  const std::size_t scratchSize = (freeSize - (freeSize % 4096)) / 2; // use half of available memory with alignment
  void *d_scratch{nullptr};
  HANDLE_CUDA_ERROR(hipMalloc(&d_scratch, scratchSize));
  std::cout << "Allocated " << scratchSize << " bytes of scratch memory on GPU\n";

  // Sphinx: MPS Marginal #8

  // Create the initial quantum state
  cutensornetState_t quantumState;
  HANDLE_CUTN_ERROR(cutensornetCreateState(cutnHandle, CUTENSORNET_STATE_PURITY_PURE, numQubits, qubitDims.data(),
                    HIP_C_64F, &quantumState));
  std::cout << "Created the initial quantum state\n";

  // Sphinx: MPS Marginal #9

  // Construct the final quantum circuit state (apply quantum gates) for the GHZ circuit
  int64_t id;
  HANDLE_CUTN_ERROR(cutensornetStateApplyTensorOperator(cutnHandle, quantumState, 1, std::vector<int32_t>{{0}}.data(),
                    d_gateH, nullptr, 1, 0, 1, &id));
  for(int32_t i = 1; i < numQubits; ++i) {
    HANDLE_CUTN_ERROR(cutensornetStateApplyTensorOperator(cutnHandle, quantumState, 2, std::vector<int32_t>{{i-1,i}}.data(),
                      d_gateCX, nullptr, 1, 0, 1, &id));
  }
  std::cout << "Applied quantum gates\n";

  // Sphinx: MPS Marginal #10

  // Specify the final target MPS representation (use default fortran strides)
  HANDLE_CUTN_ERROR(cutensornetStateFinalizeMPS(cutnHandle, quantumState, 
                    CUTENSORNET_BOUNDARY_CONDITION_OPEN, extentsPtr.data(), /*strides=*/nullptr));
  std::cout << "Requested the final MPS factorization of the quantum circuit state\n";

  // Sphinx: MPS Marginal #11

  // Optional, set up the SVD method for MPS truncation.
  cutensornetTensorSVDAlgo_t algo = CUTENSORNET_TENSOR_SVD_ALGO_GESVDJ; 
  HANDLE_CUTN_ERROR(cutensornetStateConfigure(cutnHandle, quantumState, 
                    CUTENSORNET_STATE_CONFIG_MPS_SVD_ALGO, &algo, sizeof(algo)));
  std::cout << "Configured the MPS factorization computation\n";

  // Sphinx: MPS Marginal #12

  // Prepare the MPS computation and attach workspace
  cutensornetWorkspaceDescriptor_t workDesc;
  HANDLE_CUTN_ERROR(cutensornetCreateWorkspaceDescriptor(cutnHandle, &workDesc));
  std::cout << "Created the workspace descriptor\n";
  HANDLE_CUTN_ERROR(cutensornetStatePrepare(cutnHandle, quantumState, scratchSize, workDesc, 0x0));
  std::cout << "Prepared the computation of the quantum circuit state\n";
  double flops {0.0};
  HANDLE_CUTN_ERROR(cutensornetStateGetInfo(cutnHandle, quantumState,
                    CUTENSORNET_STATE_INFO_FLOPS, &flops, sizeof(flops)));
  if(flops > 0.0) {
    std::cout << "Total flop count = " << (flops/1e9) << " GFlop\n";
  }else if(flops < 0.0) {
    std::cout << "ERROR: Negative Flop count!\n";
    std::abort();
  }

  int64_t worksize {0};
  HANDLE_CUTN_ERROR(cutensornetWorkspaceGetMemorySize(cutnHandle,
                                                      workDesc,
                                                      CUTENSORNET_WORKSIZE_PREF_RECOMMENDED,
                                                      CUTENSORNET_MEMSPACE_DEVICE,
                                                      CUTENSORNET_WORKSPACE_SCRATCH,
                                                      &worksize));
  std::cout << "Scratch GPU workspace size (bytes) for MPS computation = " << worksize << std::endl;
  if(worksize <= scratchSize) {
    HANDLE_CUTN_ERROR(cutensornetWorkspaceSetMemory(cutnHandle, workDesc, CUTENSORNET_MEMSPACE_DEVICE,
                      CUTENSORNET_WORKSPACE_SCRATCH, d_scratch, worksize));
  }else{
    std::cout << "ERROR: Insufficient workspace size on Device!\n";
    std::abort();
  }
  std::cout << "Set the workspace buffer for the MPS factorization computation\n";

  // Sphinx: MPS Marginal #13

  // Execute MPS computation
  HANDLE_CUTN_ERROR(cutensornetStateCompute(cutnHandle, quantumState, 
                    workDesc, extentsPtr.data(), /*strides=*/nullptr, d_mpsTensors.data(), 0));
  std::cout << "Computed the MPS factorization\n";

  // Sphinx: MPS Marginal #14

  // Specify the desired reduced density matrix (marginal)
  cutensornetStateMarginal_t marginal;
  HANDLE_CUTN_ERROR(cutensornetCreateMarginal(cutnHandle, quantumState, numMarginalModes, marginalModes.data(),
                    0, nullptr, std::vector<int64_t>{{1,2,4,8}}.data(), &marginal)); // using explicit strides
  std::cout << "Created the specified quantum circuit reduced densitry matrix (marginal)\n";

  // Sphinx: MPS Marginal #15

  // Configure the computation of the specified quantum circuit reduced density matrix (marginal)
  const int32_t numHyperSamples = 8; // desired number of hyper samples used in the tensor network contraction path finder
  HANDLE_CUTN_ERROR(cutensornetMarginalConfigure(cutnHandle, marginal,
                    CUTENSORNET_MARGINAL_CONFIG_NUM_HYPER_SAMPLES, &numHyperSamples, sizeof(numHyperSamples)));
  std::cout << "Configured the specified quantum circuit reduced density matrix (marginal) computation\n";

  // Sphinx: MPS Marginal #16

  // Prepare the specified quantum circuit reduced densitry matrix (marginal)
  HANDLE_CUTN_ERROR(cutensornetMarginalPrepare(cutnHandle, marginal, scratchSize, workDesc, 0x0));
  std::cout << "Prepared the specified quantum circuit reduced density matrix (marginal)\n";
  flops = 0.0;
  HANDLE_CUTN_ERROR(cutensornetMarginalGetInfo(cutnHandle, marginal,
                    CUTENSORNET_MARGINAL_INFO_FLOPS, &flops, sizeof(flops)));
  std::cout << "Total flop count = " << (flops/1e9) << " GFlop\n";
  if(flops <= 0.0) {
    std::cout << "ERROR: Invalid Flop count!\n";
    std::abort();
  }

  // Sphinx: MPS Marginal #17

  // Attach the workspace buffer
  HANDLE_CUTN_ERROR(cutensornetWorkspaceGetMemorySize(cutnHandle,
                                                      workDesc,
                                                      CUTENSORNET_WORKSIZE_PREF_RECOMMENDED,
                                                      CUTENSORNET_MEMSPACE_DEVICE,
                                                      CUTENSORNET_WORKSPACE_SCRATCH,
                                                      &worksize));
  std::cout << "Required scratch GPU workspace size (bytes) for marginal computation = " << worksize << std::endl;
  if(worksize <= scratchSize) {
    HANDLE_CUTN_ERROR(cutensornetWorkspaceSetMemory(cutnHandle, workDesc, CUTENSORNET_MEMSPACE_DEVICE,
                      CUTENSORNET_WORKSPACE_SCRATCH, d_scratch, worksize));
  }else{
    std::cout << "ERROR: Insufficient workspace size on Device!\n";
    std::abort();
  }
  std::cout << "Set the workspace buffer\n";
  
  // Sphinx: MPS Marginal #18

  // Compute the specified quantum circuit reduced densitry matrix (marginal)
  HANDLE_CUTN_ERROR(cutensornetMarginalCompute(cutnHandle, marginal, nullptr, workDesc, d_rdm, 0));
  std::cout << "Computed the specified quantum circuit reduced density matrix (marginal)\n";
  std::vector<std::complex<double>> h_rdm(rdmSize);
  HANDLE_CUDA_ERROR(hipMemcpy(h_rdm.data(), d_rdm, rdmSize * (2 * fp64size), hipMemcpyDeviceToHost));
  std::cout << "Reduced density matrix for " << numMarginalModes << " qubits:\n";
  for(std::size_t i = 0; i < rdmDim; ++i) {
    for(std::size_t j = 0; j < rdmDim; ++j) {
      std::cout << " " << h_rdm[i + j * rdmDim];
    }
    std::cout << std::endl;
  }

  // Sphinx: MPS Marginal #19

  // Destroy the workspace descriptor
  HANDLE_CUTN_ERROR(cutensornetDestroyWorkspaceDescriptor(workDesc));
  std::cout << "Destroyed the workspace descriptor\n";

  // Destroy the quantum circuit reduced density matrix
  HANDLE_CUTN_ERROR(cutensornetDestroyMarginal(marginal));
  std::cout << "Destroyed the quantum circuit state reduced density matrix (marginal)\n";

  // Destroy the quantum circuit state
  HANDLE_CUTN_ERROR(cutensornetDestroyState(quantumState));
  std::cout << "Destroyed the quantum circuit state\n";

  for (int32_t i = 0; i < numQubits; i++) {
    HANDLE_CUDA_ERROR(hipFree(d_mpsTensors[i]));
  }
  HANDLE_CUDA_ERROR(hipFree(d_scratch));
  HANDLE_CUDA_ERROR(hipFree(d_rdm));
  HANDLE_CUDA_ERROR(hipFree(d_gateCX));
  HANDLE_CUDA_ERROR(hipFree(d_gateH));
  std::cout << "Freed memory on GPU\n";

  // Finalize the cuTensorNet library
  HANDLE_CUTN_ERROR(cutensornetDestroy(cutnHandle));
  std::cout << "Finalized the cuTensorNet library\n";

  return 0;
}
