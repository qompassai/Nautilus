#include "hip/hip_runtime.h"
/* Copyright (c) 2023-2024, NVIDIA CORPORATION & AFFILIATES.
 *
 * SPDX-License-Identifier: BSD-3-Clause
 */

// Sphinx: MPS Amplitudes #1

#include <cstdlib>
#include <cstdio>
#include <cassert>
#include <complex>
#include <vector>
#include <bitset>
#include <iostream>

#include <hip/hip_runtime.h>
#include <cutensornet.h>


#define HANDLE_CUDA_ERROR(x) \
{ const auto err = x; \
  if( err != hipSuccess ) \
  { printf("CUDA error %s in line %d\n", hipGetErrorString(err), __LINE__); fflush(stdout); std::abort(); } \
};

#define HANDLE_CUTN_ERROR(x) \
{ const auto err = x; \
  if( err != CUTENSORNET_STATUS_SUCCESS ) \
  { printf("cuTensorNet error %s in line %d\n", cutensornetGetErrorString(err), __LINE__); fflush(stdout); std::abort(); } \
};


int main()
{
  static_assert(sizeof(size_t) == sizeof(int64_t), "Please build this sample on a 64-bit architecture!");

  constexpr std::size_t fp64size = sizeof(double);

  // Sphinx: MPS Amplitudes #2

  // Quantum state configuration
  constexpr int32_t numQubits = 6; // number of qubits
  const std::vector<int64_t> qubitDims(numQubits,2); // qubit dimensions
  const std::vector<int32_t> fixedModes({0,1}); // fixed modes in the output amplitude tensor (must be in acsending order)
  const std::vector<int64_t> fixedValues({1,1}); // values of the fixed modes in the output amplitude tensor
  const int32_t numFixedModes = fixedModes.size(); // number of fixed modes in the output amplitude tensor
  std::cout << "Quantum circuit: " << numQubits << " qubits\n";

  // Sphinx: MPS Amplitudes #3

  // Initialize the cuTensorNet library
  HANDLE_CUDA_ERROR(hipSetDevice(0));
  cutensornetHandle_t cutnHandle;
  HANDLE_CUTN_ERROR(cutensornetCreate(&cutnHandle));
  std::cout << "Initialized cuTensorNet library on GPU 0\n";

  // Sphinx: MPS Amplitudes #4

  // Define necessary quantum gate tensors in Host memory
  const double invsq2 = 1.0 / std::sqrt(2.0);
  //  Hadamard gate
  const std::vector<std::complex<double>> h_gateH {{invsq2, 0.0},  {invsq2, 0.0},
                                                   {invsq2, 0.0}, {-invsq2, 0.0}};
  //  CX gate
  const std::vector<std::complex<double>> h_gateCX {{1.0, 0.0}, {0.0, 0.0}, {0.0, 0.0}, {0.0, 0.0},
                                                    {0.0, 0.0}, {1.0, 0.0}, {0.0, 0.0}, {0.0, 0.0},
                                                    {0.0, 0.0}, {0.0, 0.0}, {0.0, 0.0}, {1.0, 0.0},
                                                    {0.0, 0.0}, {0.0, 0.0}, {1.0, 0.0}, {0.0, 0.0}};

  // Copy quantum gates to Device memory
  void *d_gateH{nullptr}, *d_gateCX{nullptr};
  HANDLE_CUDA_ERROR(hipMalloc(&d_gateH, 4 * (2 * fp64size)));
  HANDLE_CUDA_ERROR(hipMalloc(&d_gateCX, 16 * (2 * fp64size)));
  std::cout << "Allocated quantum gate memory on GPU\n";
  HANDLE_CUDA_ERROR(hipMemcpy(d_gateH, h_gateH.data(), 4 * (2 * fp64size), hipMemcpyHostToDevice));
  HANDLE_CUDA_ERROR(hipMemcpy(d_gateCX, h_gateCX.data(), 16 * (2 * fp64size), hipMemcpyHostToDevice));
  std::cout << "Copied quantum gates to GPU memory\n";

  // Sphinx: MPS Amplitudes #5

  // Determine the MPS representation and allocate buffers for the MPS tensors
  const int64_t maxExtent = 2; // GHZ state can be exactly represented with max bond dimension of 2
  std::vector<std::vector<int64_t>> extents;
  std::vector<int64_t*> extentsPtr(numQubits); 
  std::vector<void*> d_mpsTensors(numQubits, nullptr);
  for (int32_t i = 0; i < numQubits; i++) {
    if (i == 0) { // left boundary MPS tensor
      extents.push_back({2, maxExtent});
      HANDLE_CUDA_ERROR(hipMalloc(&d_mpsTensors[i], 2 * maxExtent * 2 * fp64size));
    }
    else if (i == numQubits-1) { // right boundary MPS tensor
      extents.push_back({maxExtent, 2});
      HANDLE_CUDA_ERROR(hipMalloc(&d_mpsTensors[i], 2 * maxExtent * 2 * fp64size));
    }
    else { // middle MPS tensors
      extents.push_back({maxExtent, 2, maxExtent});
      HANDLE_CUDA_ERROR(hipMalloc(&d_mpsTensors[i], 2 * maxExtent * maxExtent * 2 * fp64size));
    }
    extentsPtr[i] = extents[i].data();
  }

  // Sphinx: MPS Amplitudes #6

  // Allocate Device memory for the specified slice of the quantum circuit amplitudes tensor
  void *d_amp{nullptr};
  std::size_t ampSize = 1;
  for(const auto & qubitDim: qubitDims) ampSize *= qubitDim; // all state modes (full size)
  for(const auto & fixedMode: fixedModes) ampSize /= qubitDims[fixedMode]; // fixed state modes reduce the slice size
  HANDLE_CUDA_ERROR(hipMalloc(&d_amp, ampSize * (2 * fp64size)));
  std::cout << "Allocated memory for the specified slice of the quantum circuit amplitude tensor of size "
            << ampSize << " elements\n";

  // Sphinx: MPS Amplitudes #7

  // Query the free memory on Device
  std::size_t freeSize{0}, totalSize{0};
  HANDLE_CUDA_ERROR(hipMemGetInfo(&freeSize, &totalSize));
  const std::size_t scratchSize = (freeSize - (freeSize % 4096)) / 2; // use half of available memory with alignment
  void *d_scratch{nullptr};
  HANDLE_CUDA_ERROR(hipMalloc(&d_scratch, scratchSize));
  std::cout << "Allocated " << scratchSize << " bytes of scratch memory on GPU\n";

  // Sphinx: MPS Amplitudes #8

  // Create the initial quantum state
  cutensornetState_t quantumState;
  HANDLE_CUTN_ERROR(cutensornetCreateState(cutnHandle, CUTENSORNET_STATE_PURITY_PURE, numQubits, qubitDims.data(),
                    HIP_C_64F, &quantumState));
  std::cout << "Created the initial quantum state\n";

  // Sphinx: MPS Amplitudes #9

  // Construct the final quantum circuit state (apply quantum gates) for the GHZ circuit
  int64_t id;
  HANDLE_CUTN_ERROR(cutensornetStateApplyTensorOperator(cutnHandle, quantumState, 1, std::vector<int32_t>{{0}}.data(),
                    d_gateH, nullptr, 1, 0, 1, &id));
  for(int32_t i = 1; i < numQubits; ++i) {
    HANDLE_CUTN_ERROR(cutensornetStateApplyTensorOperator(cutnHandle, quantumState, 2, std::vector<int32_t>{{i-1,i}}.data(),
                      d_gateCX, nullptr, 1, 0, 1, &id));
  }
  std::cout << "Applied quantum gates\n";

  // Sphinx: MPS Amplitudes #10

  // Specify the final target MPS representation (use default fortran strides)
  HANDLE_CUTN_ERROR(cutensornetStateFinalizeMPS(cutnHandle, quantumState, 
                    CUTENSORNET_BOUNDARY_CONDITION_OPEN, extentsPtr.data(), /*strides=*/nullptr));
  std::cout << "Requested the final MPS factorization of the quantum circuit state\n";

  // Sphinx: MPS Amplitudes #11

  // Optional, set up the SVD method for MPS truncation.
  cutensornetTensorSVDAlgo_t algo = CUTENSORNET_TENSOR_SVD_ALGO_GESVDJ; 
  HANDLE_CUTN_ERROR(cutensornetStateConfigure(cutnHandle, quantumState, 
                    CUTENSORNET_STATE_CONFIG_MPS_SVD_ALGO, &algo, sizeof(algo)));
  std::cout << "Configured the MPS factorization computation\n";

  // Sphinx: MPS Amplitudes #12

  // Prepare the MPS computation and attach workspace
  cutensornetWorkspaceDescriptor_t workDesc;
  HANDLE_CUTN_ERROR(cutensornetCreateWorkspaceDescriptor(cutnHandle, &workDesc));
  std::cout << "Created the workspace descriptor\n";
  HANDLE_CUTN_ERROR(cutensornetStatePrepare(cutnHandle, quantumState, scratchSize, workDesc, 0x0));
  std::cout << "Prepared the computation of the quantum circuit state\n";
  double flops {0.0};
  HANDLE_CUTN_ERROR(cutensornetStateGetInfo(cutnHandle, quantumState,
                    CUTENSORNET_STATE_INFO_FLOPS, &flops, sizeof(flops)));
  if(flops > 0.0) {
    std::cout << "Total flop count = " << (flops/1e9) << " GFlop\n";
  }else if(flops < 0.0) {
    std::cout << "ERROR: Negative Flop count!\n";
    std::abort();
  }

  int64_t worksize {0};
  HANDLE_CUTN_ERROR(cutensornetWorkspaceGetMemorySize(cutnHandle,
                                                      workDesc,
                                                      CUTENSORNET_WORKSIZE_PREF_RECOMMENDED,
                                                      CUTENSORNET_MEMSPACE_DEVICE,
                                                      CUTENSORNET_WORKSPACE_SCRATCH,
                                                      &worksize));
  std::cout << "Scratch GPU workspace size (bytes) for MPS computation = " << worksize << std::endl;
  if(worksize <= scratchSize) {
    HANDLE_CUTN_ERROR(cutensornetWorkspaceSetMemory(cutnHandle, workDesc, CUTENSORNET_MEMSPACE_DEVICE,
                      CUTENSORNET_WORKSPACE_SCRATCH, d_scratch, worksize));
  }else{
    std::cout << "ERROR: Insufficient workspace size on Device!\n";
    std::abort();
  }
  std::cout << "Set the workspace buffer for the MPS factorization computation\n";

  // Sphinx: MPS Amplitudes #13

  // Execute MPS computation
  HANDLE_CUTN_ERROR(cutensornetStateCompute(cutnHandle, quantumState, 
                    workDesc, extentsPtr.data(), /*strides=*/nullptr, d_mpsTensors.data(), 0));
  std::cout << "Computed the MPS factorization\n";

  // Sphinx: MPS Amplitudes #14

  // Specify the quantum circuit amplitudes accessor
  cutensornetStateAccessor_t accessor;
  HANDLE_CUTN_ERROR(cutensornetCreateAccessor(cutnHandle, quantumState, numFixedModes, fixedModes.data(),
                    nullptr, &accessor)); // using default strides
  std::cout << "Created the specified quantum circuit amplitudes accessor\n";

  // Sphinx: MPS Amplitudes #15

  // Configure the computation of the slice of the specified quantum circuit amplitudes tensor
  const int32_t numHyperSamples = 8; // desired number of hyper samples used in the tensor network contraction path finder
  HANDLE_CUTN_ERROR(cutensornetAccessorConfigure(cutnHandle, accessor,
                    CUTENSORNET_ACCESSOR_CONFIG_NUM_HYPER_SAMPLES, &numHyperSamples, sizeof(numHyperSamples)));

  // Sphinx: MPS Amplitudes #16

  // Prepare the computation of the specified slice of the quantum circuit amplitudes tensor
  HANDLE_CUTN_ERROR(cutensornetAccessorPrepare(cutnHandle, accessor, scratchSize, workDesc, 0x0));
  std::cout << "Prepared the computation of the specified slice of the quantum circuit amplitudes tensor\n";
  flops = 0.0;
  HANDLE_CUTN_ERROR(cutensornetAccessorGetInfo(cutnHandle, accessor,
                    CUTENSORNET_ACCESSOR_INFO_FLOPS, &flops, sizeof(flops)));
  std::cout << "Total flop count = " << (flops/1e9) << " GFlop\n";
  if(flops <= 0.0) {
    std::cout << "ERROR: Invalid Flop count!\n";
    std::abort();
  }

  // Sphinx: MPS Amplitudes #17

  // Attach the workspace buffer
  HANDLE_CUTN_ERROR(cutensornetWorkspaceGetMemorySize(cutnHandle,
                                                      workDesc,
                                                      CUTENSORNET_WORKSIZE_PREF_RECOMMENDED,
                                                      CUTENSORNET_MEMSPACE_DEVICE,
                                                      CUTENSORNET_WORKSPACE_SCRATCH,
                                                      &worksize));
  std::cout << "Required scratch GPU workspace size (bytes) = " << worksize << std::endl;
  if(worksize <= scratchSize) {
    HANDLE_CUTN_ERROR(cutensornetWorkspaceSetMemory(cutnHandle, workDesc, CUTENSORNET_MEMSPACE_DEVICE,
                      CUTENSORNET_WORKSPACE_SCRATCH, d_scratch, worksize));
  }else{
    std::cout << "ERROR: Insufficient workspace size on Device!\n";
    std::abort();
  }
  std::cout << "Set the workspace buffer\n";

  // Sphinx: MPS Amplitudes #18

  // Compute the specified slice of the quantum circuit amplitudes tensor
  std::complex<double> stateNorm2{0.0,0.0};
  HANDLE_CUTN_ERROR(cutensornetAccessorCompute(cutnHandle, accessor, fixedValues.data(),
                    workDesc, d_amp, static_cast<void*>(&stateNorm2), 0x0));
  std::cout << "Computed the specified slice of the quantum circuit amplitudes tensor\n";
  std::vector<std::complex<double>> h_amp(ampSize);
  HANDLE_CUDA_ERROR(hipMemcpy(h_amp.data(), d_amp, ampSize * (2 * fp64size), hipMemcpyDeviceToHost));
  std::cout << "Amplitudes slice for " << (numQubits - numFixedModes) << " qubits:\n";
  for(std::size_t i = 0; i < ampSize; ++i) {
    std::cout << " " << h_amp[i] << std::endl;
  }
  std::cout << "Squared 2-norm of the state = (" << stateNorm2.real() << ", " << stateNorm2.imag() << ")\n";

  // Sphinx: MPS Amplitudes #19

  // Destroy the workspace descriptor
  HANDLE_CUTN_ERROR(cutensornetDestroyWorkspaceDescriptor(workDesc));
  std::cout << "Destroyed the workspace descriptor\n";

  // Destroy the quantum circuit amplitudes accessor
  HANDLE_CUTN_ERROR(cutensornetDestroyAccessor(accessor));
  std::cout << "Destroyed the quantum circuit amplitudes accessor\n";

  // Destroy the quantum circuit state
  HANDLE_CUTN_ERROR(cutensornetDestroyState(quantumState));
  std::cout << "Destroyed the quantum circuit state\n";

  for (int32_t i = 0; i < numQubits; i++) {
    HANDLE_CUDA_ERROR(hipFree(d_mpsTensors[i]));
  }
  HANDLE_CUDA_ERROR(hipFree(d_scratch));
  HANDLE_CUDA_ERROR(hipFree(d_amp));
  HANDLE_CUDA_ERROR(hipFree(d_gateCX));
  HANDLE_CUDA_ERROR(hipFree(d_gateH));
  std::cout << "Freed memory on GPU\n";

  // Finalize the cuTensorNet library
  HANDLE_CUTN_ERROR(cutensornetDestroy(cutnHandle));
  std::cout << "Finalized the cuTensorNet library\n";

  return 0;
}
